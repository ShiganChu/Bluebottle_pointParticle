#include "hip/hip_runtime.h"
__global__ void lpt_mollify_momentum_z(int npoints,real *epsp, real *fz, point_struct *points, dom_struct *dom, real dt)
{

int index =  threadIdx.x + blockIdx.x*blockDim.x;

if(index<1)
{
//Define the stencil of the gausian filter
real ksi[STENCIL][STENCIL][STENCIL];
real xs=dom->xs;real ys=dom->ys;real zs=dom->zs;
real dx=dom->dx;real dy=dom->dy;real dz=dom->dz;
  real ddx = 1. / dom->dx;
  real ddy = 1. / dom->dy;
  real ddz = 1. / dom->dz;
//real cellVol=dx*dy*dz;
for(int pp=0;pp<npoints;pp++)
{

real  xp =  points[pp].x;
real  yp =  points[pp].y;
real  zp =  points[pp].z;

real rad=points[pp].r;
real Vp=PI*4/3*rad*rad*rad;
real mp=Vp*points[pp].rho + points[pp].ms;
//ms*w+m*du*dt
real srcW=-(points[pp].msdot*points[pp].w+points[pp].wdot*dt*mp);

  int ip,jp,kp;
  // interpolate w-momentum source
  ip = round((x - dom->xs) * ddx) + DOM_BUF;
  jp = floor((y - dom->ys) * ddy) + DOM_BUF;
  kp = floor((z - dom->zs) * ddz - 0.5) + DOM_BUF;

real buf=0;
int began=-(STENCIL-1)/2;//equal to -1 if STENCIL=3
int end=1+(STENCIL-1)/2;//equal to 2 if STENCIL=3

for(int dk=began;dk<end;dk++)
for(int dj=began;dj<end;dj++)
for(int di=began;di<end;di++)
{
ksi[di-began][dj-began][dk-began]=lpt_integrate_mol_w(ip+di,jp+dj,kp+dk,xp,yp,zp,dx,dy,dz,xs,ys,zs);
buf+=ksi[di-began][dj-began][dk-began];
}


// Normalize  ksi = ksi/buf
//if(fabs(buf-1)>EPSILON)
if(buf>0)
{
for(int dk=began;dk<end;dk++)
for(int dj=began;dj<end;dj++)
for(int di=began;di<end;di++)
{ksi[di-began][dj-began][dk-began]=ksi[di-began][dj-began][dk-began]/buf;}
}


int ic,jc,kc;
// Perform the actual extrapolation on A
for(int dk=began;dk<end;dk++)
for(int dj=began;dj<end;dj++)
for(int di=began;di<end;di++)
{
//TODO make sure the scale is correct
//A[(di+ip)+(dj+jp)*dom->Gfz._s1b+(dk+kp)*dom->Gfz._s2b]+=ksi[di-began][dj-began][dk-began]*Ap*pointPartVol/epsp/domVol;
  ic=di+ip;jc=dj+jp;kc=dk+kp;
  if(ic < dom->Gfz.is) ic = ic+(dom->Gfz.in-1);
  if(jc < dom->Gfz.js) jc = jc+dom->Gfz.jn;
  if(kc < dom->Gfz.ks) kc = kc+dom->Gfz.kn;
  if(ic > dom->Gfz.ie-1) ic = ic-(dom->Gfz.in-1);
  if(jc > dom->Gfz.je-1) jc = jc-dom->Gfz.jn;
  if(kc > dom->Gfz.ke-1) kc = kc-dom->Gfz.kn;
fz[ic+jc*dom->Gfz._s1b+kc*dom->Gfz._s2b]+=ksi[di-began][dj-began][dk-began]*srcW;
}

}
}
}

