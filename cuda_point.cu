#include "hip/hip_runtime.h"
//#include "cuda_point.h"
//#include "point.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

//#include "bluebottle.h"
#include "cuda_point.h"

extern "C"
void cuda_flow_stress()
{
  #pragma omp parallel num_threads(nsubdom)
  {
    int dev = omp_get_thread_num();
    checkCudaErrors(hipSetDevice(dev + dev_start));
//printf("\ndev in cuda_flow_stress %d %d\n",dev,dev_start);
    int threads_x = 0;
    int threads_y = 0;
    int threads_z = 0;
    int blocks_x = 0;
    int blocks_y = 0;
    int blocks_z = 0;

   // u-component
    if(dom[dev].Gfx.jnb < MAX_THREADS_DIM)
      threads_y = dom[dev].Gfx.jnb + 2;
    else
      threads_y = MAX_THREADS_DIM;

    if(dom[dev].Gfx.knb < MAX_THREADS_DIM)
      threads_z = dom[dev].Gfx.knb + 2;
    else
      threads_z = MAX_THREADS_DIM;

    blocks_y = (int)ceil((real) dom[dev].Gfx.jnb / (real) (threads_y-2));
    blocks_z = (int)ceil((real) dom[dev].Gfx.knb / (real) (threads_z-2));



    dim3 dimBlocks_u(threads_y, threads_z);
    dim3 numBlocks_u(blocks_y, blocks_z);
    stress_u<<<numBlocks_u, dimBlocks_u>>>(rho_f, nu,_u[dev],_p[dev],_p0[dev], _stress_u[dev], _dom[dev],_flag_u[dev],dt,dt0);
fflush(stdout);



// v-component
    if(dom[dev].Gfy.knb < MAX_THREADS_DIM)
      threads_z = dom[dev].Gfy.knb + 2;
    else
      threads_z = MAX_THREADS_DIM;

    if(dom[dev].Gfy.inb < MAX_THREADS_DIM)
      threads_x = dom[dev].Gfy.inb + 2;
    else
      threads_x = MAX_THREADS_DIM;
  
    blocks_z = (int)ceil((real) dom[dev].Gfy.knb / (real) (threads_z-2));
    blocks_x = (int)ceil((real) dom[dev].Gfy.inb / (real) (threads_x-2));

    dim3 dimBlocks_v(threads_z, threads_x);
    dim3 numBlocks_v(blocks_z, blocks_x);

    stress_v<<<numBlocks_v, dimBlocks_v>>>(rho_f, nu,_v[dev],_p[dev],_p0[dev], _stress_v[dev], _dom[dev],_flag_v[dev],dt,dt0);
fflush(stdout);

// w-component
    if(dom[dev].Gfz.inb < MAX_THREADS_DIM)
      threads_x = dom[dev].Gfz.inb + 2;
    else
      threads_x = MAX_THREADS_DIM;

    if(dom[dev].Gfz.jnb < MAX_THREADS_DIM)
      threads_y = dom[dev].Gfz.jnb + 2;
    else
      threads_y = MAX_THREADS_DIM;

    blocks_x = (int)ceil((real) dom[dev].Gfz.inb / (real) (threads_x-2));
    blocks_y = (int)ceil((real) dom[dev].Gfz.jnb / (real) (threads_y-2));

    dim3 dimBlocks_w(threads_x, threads_y);
    dim3 numBlocks_w(blocks_x, blocks_y);

    stress_w<<<numBlocks_w, dimBlocks_w>>>(rho_f, nu,_w[dev],_p[dev],_p0[dev], _stress_w[dev], _dom[dev],_flag_w[dev],dt,dt0);
fflush(stdout);
 }

}


extern "C"
void cuda_move_points()
{



   // parallelize over CPU threads
  #pragma omp parallel num_threads(nsubdom)
  {
    int dev = omp_get_thread_num();
    checkCudaErrors(hipSetDevice(dev + dev_start));
/*
    int threads = MAX_THREADS_1D;
    int blocks = (int)ceil((real) npoints / (real) threads);

    dim3 dimBlocks(threads);
    dim3 numBlocks(blocks);
*/

    dim3 dimBlocks_p;
    dim3 numBlocks_p;
    block_thread_point(dimBlocks_p,numBlocks_p,npoints);

    
    if(npoints > 0) {
      // do collision forcing
      /** if there are n point_particles in a close group, repeat this n times **/
    /*
      real *forces;
      checkCudaErrors(hipMalloc((void**) &forces, 3*npoints*sizeof(real)));
      gpumem += 3 * npoints * sizeof(real);
      real *moments;
      checkCudaErrors(hipMalloc((void**) &moments, 3*npoints*sizeof(real)));
      gpumem += 3 * npoints * sizeof(real);
      real eps = 0.1 * (Dom.dx + Dom.dy + Dom.dz) / 3.;
*/


//for(int l = 0; l < 10; l++) {
   //   collision_init<<<numBlocks_p, dimBlocks_p>>>(_points[dev], npoints);
    
 /*
       for(int i = 0; i < npoints; i++) {
          collision_points<<<numBlocks_p, dimBlocks_p>>>(_points[dev], i,
            _dom[dev], eps, forces, moments, npoints, mu, bc);
        }
        spring_points<<<numBlocks_p, dimBlocks_p>>>(_points[dev], npoints);
        collision_walls<<<numBlocks_p, dimBlocks_p>>>(_dom[dev], _points[dev],
          npoints, bc, eps, mu);
      }
   */



//bc is bc.uTD etc. Make sure which BC this is. 
point_interp_init<<<numBlocks_p, dimBlocks_p>>>(npoints,_points[dev],
						ug[dev],vg[dev],wg[dev],
						lpt_stress_u[dev],lpt_stress_v[dev],lpt_stress_w[dev],scg[dev]);
fflush(stdout);

interpolate_point_vel_Lag2<<<numBlocks_p, dimBlocks_p>>>(_u[dev],_v[dev],_w[dev],
							 npoints,rho_f,nu,
							 ug[dev],vg[dev],wg[dev],
							_points[dev],_dom[dev],bc);
fflush(stdout);

interpolate_point_scalar_Lag2<<<numBlocks_p, dimBlocks_p>>>(npoints,_sc[dev],scg[dev],_points[dev],_dom[dev]);
fflush(stdout);



/*
C_add=0.5;
C_stress=1;
C_drag=1;
*/

//get lpt_stress
//TODO _stress_u is not available near the boundary(set to 0), while lpt_stress can be interpolated on BC
if(C_stress>0||C_add>0) 
interpolate_point_vel_Lag2<<<numBlocks_p, dimBlocks_p>>>(_stress_u[dev],_stress_v[dev],_stress_w[dev],
							 npoints, rho_f, nu,
							 lpt_stress_u[dev],lpt_stress_v[dev],lpt_stress_w[dev],
							 _points[dev],_dom[dev],bc);


drag_points<<<numBlocks_p, dimBlocks_p>>>(_points[dev],npoints,
ug[dev],vg[dev],wg[dev],
lpt_stress_u[dev],lpt_stress_v[dev],lpt_stress_w[dev],scg[dev],
rho_f,mu,g,gradP,
C_add, C_stress,C_drag,
sc_eq,DIFF);
fflush(stdout);

      move_points_a<<<numBlocks_p, dimBlocks_p>>>(_points[dev], npoints,dt_try);

drag_points<<<numBlocks_p, dimBlocks_p>>>(_points[dev],npoints,
ug[dev],vg[dev],wg[dev],
lpt_stress_u[dev],lpt_stress_v[dev],lpt_stress_w[dev],scg[dev],
rho_f,mu,g,gradP,
C_add, C_stress,C_drag,
sc_eq,DIFF);
fflush(stdout);

      move_points_b<<<numBlocks_p, dimBlocks_p>>>(_dom[dev], _points[dev], npoints,dt_try);

fflush(stdout);




 /*
      checkCudaErrors(hipFree(forces));
      checkCudaErrors(hipFree(moments));
 */
   		}
  	
    }
}



extern "C"
void lpt_point_twoway_forcing()
{
 // parallelize over CPU threads
  #pragma omp parallel num_threads(nsubdom)
  {
    int dev = omp_get_thread_num();
    checkCudaErrors(hipSetDevice(dev + dev_start));

int valType=0;
lpt_mollify_scH(1,valType,dev,_f_x[dev]);
lpt_mollify_scH(2,valType,dev,_f_y[dev]);
lpt_mollify_scH(3,valType,dev,_f_z[dev]);

     }
}


   













//extern "C"
void sortParticles(int *dgridParticleHash, int *dgridParticleIndex, int numParticles)
    {
//sort dgridParticleIndex based on dgridParticleHash, and they both change order
        thrust::sort_by_key(thrust::device_ptr<int>(dgridParticleHash),
                            thrust::device_ptr<int>(dgridParticleHash + numParticles),
                            thrust::device_ptr<int>(dgridParticleIndex));
    }


//The difference in mollify is the direction and mollified data, and they have different Ksi and gridParticleHash
extern "C"
void lpt_mollify_scH(int coordiSys,int valType,int dev,real *scSrc)
{

    dim3 dimBlocks_u,dimBlocks_p;
    dim3 numBlocks_u,numBlocks_p;
    dim3 numBlocks_st;
//    dim3 numBlocks_print,dimBlocks_print;

//int coordiSys=0;//coordinate systerm, cell-center or face center
int planeDirc=3;//parallel x-y or y-z or x-z plane

block_thread_cell(dimBlocks_u,numBlocks_u,dom[dev],coordiSys,planeDirc);
block_thread_point(dimBlocks_p,numBlocks_p,npoints);
block_thread_point(dimBlocks_p,numBlocks_st,npoints*STENCIL3);


int lenCell;
switch(coordiSys)
{
case 0:lenCell=dom[dev].Gcc.s3;break;
case 1:lenCell=dom[dev].Gfx.s3;break;
case 2:lenCell=dom[dev].Gfy.s3;break;
case 3:lenCell=dom[dev].Gfz.s3;break;
default: break;
}

//block_thread_point(dimBlocks_print,numBlocks_print,lenCell);

checkCudaErrors(hipMemset(gridParticleHash[dev],-1,npoints*sizeof(int)));
checkCudaErrors(hipMemset(gridParticleIndex[dev],-1,npoints*sizeof(int)));

checkCudaErrors(hipMemset(cellStart[dev],-1,lenCell*sizeof(int)));
checkCudaErrors(hipMemset(cellEnd[dev],-1,lenCell*sizeof(int)));


array_init<<<numBlocks_st, dimBlocks_p>>>(Ksi[dev],_dom[dev],npoints*STENCIL3, 0.);


calcHashD<<<numBlocks_p,dimBlocks_p>>>(gridParticleHash[dev],gridParticleIndex[dev],_points[dev],_dom[dev],npoints,coordiSys);

fflush(stdout);

sortParticles(gridParticleHash[dev],gridParticleIndex[dev],npoints);

findCellStartD<<<numBlocks_p,dimBlocks_p>>>(cellStart[dev],cellEnd[dev],gridParticleHash[dev],gridParticleIndex[dev],npoints);


//particle volume fraction 1 or other cell-centerred parameter 0
lpt_point_ksi<<<numBlocks_p,dimBlocks_p>>>(_points[dev],_dom[dev],Ksi[dev],gridParticleIndex[dev],npoints,coordiSys,valType);
fflush(stdout);

lpt_mollify_scD<<<numBlocks_u,dimBlocks_u>>>(_points[dev],_dom[dev],scSrc,Ksi[dev],cellStart[dev],cellEnd[dev],gridParticleIndex[dev],npoints,coordiSys,valType);
fflush(stdout);

//print_kernel_array_int<<<numBlocks_print,dimBlocks_print>>>(cellEnd[dev],lenCell);
}


//About Swap, and reference, dirc is the system direction, dirc2 is the plane direction
void block_thread_cell(dim3 &dimBlocks,dim3 &numBlocks,dom_struct dom,int dirc,int dirc2)
{

    int threads_y = 0;
    int threads_x = 0;
    int blocks_y = 0;
    int blocks_x = 0;

    int lenX=0;
    int lenY=0;

grid_info G;
switch(dirc)
{
case 0:G=dom.Gcc;break;
case 1:G=dom.Gfx;break;
case 2:G=dom.Gfy;break;
case 3:G=dom.Gfz;break;
default: break;
}

	switch(dirc2)
	{
	case 1:
		lenX=G._jnb;
		lenY=G._knb;
		break;
	case 2:
		lenX=G._knb;
		lenY=G._inb;
		break;
	case 3:
		lenX=G._inb;
		lenY=G._jnb;
		break;
	default: break;	
	}


    if(lenX < MAX_THREADS_DIM)
      threads_x = lenX+2;
    else
      threads_x = MAX_THREADS_DIM;

    if(lenY < MAX_THREADS_DIM)
      threads_y = lenY+2;
    else
      threads_y = MAX_THREADS_DIM;

    blocks_x = (int)ceil((real) lenX / (real) (threads_x-2));
    blocks_y = (int)ceil((real) lenY / (real) (threads_y-2));

    dimBlocks.x=threads_x;
    dimBlocks.y=threads_y;
    numBlocks.x=blocks_x;
    numBlocks.y=blocks_y;

}

void block_thread_point(dim3 &dimBlocks,dim3 &numBlocks,int npoints)
{
    int threads = MAX_THREADS_1D;
    int blocks = (int)ceil((real) npoints / (real) threads);

    dimBlocks.x=threads;
    numBlocks.x=blocks;

}


extern "C"
void cuda_malloc_array_int(int **&A,int lenArray)
{
A= (int**) malloc(nsubdom * sizeof(int*));
          cpumem += nsubdom * sizeof(int*);

  #pragma omp parallel num_threads(nsubdom)
  {
    int dev = omp_get_thread_num();
    checkCudaErrors(hipSetDevice(dev + dev_start));

   checkCudaErrors(hipMalloc((void**) &(A[dev]), sizeof(int) * lenArray));
    gpumem += sizeof(int) * lenArray;
  }
}

extern "C"
void cuda_malloc_array_real(real (**&A),int lenArray)
{   
A= (real**) malloc(nsubdom * sizeof(real*));
          cpumem += nsubdom * sizeof(real*);
      
  #pragma omp parallel num_threads(nsubdom)
  {
    int dev = omp_get_thread_num();
    checkCudaErrors(hipSetDevice(dev + dev_start));
    

   checkCudaErrors(hipMalloc((void**) &(A[dev]), sizeof(real)*lenArray));
    gpumem += sizeof(real) * lenArray;
  } 
}   


extern "C"
void cuda_free_array_real(real **&A)
{
  // free device memory on device
  #pragma omp parallel num_threads(nsubdom)
  {
    int dev = omp_get_thread_num();
    checkCudaErrors(hipSetDevice(dev + dev_start));
    checkCudaErrors(hipFree(A[dev]));
}

  free(A);

}

extern "C"
void cuda_free_array_int(int **&A)
{
  // free device memory on device
  #pragma omp parallel num_threads(nsubdom)
  {
    int dev = omp_get_thread_num();
    checkCudaErrors(hipSetDevice(dev + dev_start));

    checkCudaErrors(hipFree(A[dev]));
}   
    
  free(A);

}

extern "C"
void cuda_point_malloc(void)
{
  // allocate device memory on host
  _points = (point_struct**) malloc(nsubdom * sizeof(point_struct*));
  cpumem += nsubdom * sizeof(point_struct*);

  _flag_u = (int**) malloc(nsubdom * sizeof(int*));
  cpumem += nsubdom * sizeof(int*);
  _flag_v = (int**) malloc(nsubdom * sizeof(int*));
  cpumem += nsubdom * sizeof(int*);
  _flag_w = (int**) malloc(nsubdom * sizeof(int*));
  cpumem += nsubdom * sizeof(int*);

  // allocate device memory on device
  #pragma omp parallel num_threads(nsubdom)
  {
    int dev = omp_get_thread_num();
    checkCudaErrors(hipSetDevice(dev + dev_start));

    checkCudaErrors(hipMalloc((void**) &(_points[dev]),
      sizeof(point_struct) * npoints));
    gpumem += sizeof(point_struct) * npoints;

   checkCudaErrors(hipMalloc((void**) &(_flag_u[dev]),
      sizeof(int) * dom[dev].Gfx.s3b));
    gpumem += sizeof(int) * dom[dev].Gfx.s3b;
    checkCudaErrors(hipMalloc((void**) &(_flag_v[dev]),
      sizeof(int) * dom[dev].Gfy.s3b));
    gpumem += sizeof(int) * dom[dev].Gfy.s3b;
    checkCudaErrors(hipMalloc((void**) &(_flag_w[dev]),
      sizeof(int) * dom[dev].Gfz.s3b));
    gpumem += sizeof(int) * dom[dev].Gfz.s3b;

  }

cuda_malloc_array_real(ug,npoints);
cuda_malloc_array_real(vg,npoints);
cuda_malloc_array_real(wg,npoints);

cuda_malloc_array_real(lpt_stress_u,npoints);
cuda_malloc_array_real(lpt_stress_v,npoints);
cuda_malloc_array_real(lpt_stress_w,npoints);

cuda_malloc_array_real(scg,npoints);
cuda_malloc_array_real(Ksi,npoints*STENCIL3);

cuda_malloc_array_int(gridParticleIndex,npoints);
cuda_malloc_array_int(gridParticleHash,npoints);

//calculate the maximum length of coordinate system
int len0=dom[0].Gcc.s3b;
int len1=dom[0].Gfx.s3b;
int len2=dom[0].Gfy.s3b;
int len3=dom[0].Gfz.s3b;
if(len0<len1) len0=len1;
if(len2<len3) len2=len3;
if(len0<len2) len0=len2;

cuda_malloc_array_int(cellStart,len0);
cuda_malloc_array_int(cellEnd,len0);

}

extern "C"
void cuda_point_free(void)
{
  // free device memory on device
  #pragma omp parallel num_threads(nsubdom)
  {
    int dev = omp_get_thread_num();
    checkCudaErrors(hipSetDevice(dev + dev_start));

    checkCudaErrors(hipFree(_points[dev]));
    checkCudaErrors(hipFree(_flag_u[dev]));
    checkCudaErrors(hipFree(_flag_v[dev]));
    checkCudaErrors(hipFree(_flag_w[dev]));
  

}

cuda_free_array_real(ug);
cuda_free_array_real(vg);
cuda_free_array_real(wg);

cuda_free_array_real(scg);
cuda_free_array_real(Ksi);

cuda_free_array_real(lpt_stress_u);
cuda_free_array_real(lpt_stress_v);
cuda_free_array_real(lpt_stress_w);

cuda_free_array_int(cellStart);
cuda_free_array_int(cellEnd);
cuda_free_array_int(gridParticleIndex);
cuda_free_array_int(gridParticleHash);


  free(_points);
  free(_flag_u);
  free(_flag_v);
  free(_flag_w);


}

extern "C"
void cuda_point_push(void)
{
  // copy host data to device
  #pragma omp parallel num_threads(nsubdom)
  {
    int dev = omp_get_thread_num();
    checkCudaErrors(hipSetDevice(dev + dev_start));

    checkCudaErrors(hipMemcpy(_points[dev], points, sizeof(point_struct) * npoints,hipMemcpyHostToDevice));
  //  checkCudaErrors(hipMemcpy(points,_points[dev],sizeof(point_struct) * npoints,hipMemcpyDeviceToHost));
 }
}

extern "C"
void cuda_point_pull(void)
{
  // all devices have the same point_particle data for now, so just copy one of them
  checkCudaErrors(hipMemcpy(points, _points[0], sizeof(point_struct) * npoints,
    hipMemcpyDeviceToHost));

}



extern "C"
void cuda_build_cages(void)
{
  cuda_point_pull();

  // parallelize over domains
  #pragma omp parallel num_threads(nsubdom)
  {
    int dev = omp_get_thread_num();
    checkCudaErrors(hipSetDevice(dev + dev_start));

    int threads_x = 0;
    int threads_y = 0;
    int threads_z = 0;
    int blocks_x = 0;
    int blocks_y = 0;
    int blocks_z = 0;


    // reset flag_u
    if(dom[dev].Gfx.jn < MAX_THREADS_DIM)
      threads_y = dom[dev].Gfx.jnb;
    else
      threads_y = MAX_THREADS_DIM;

    if(dom[dev].Gfx.kn < MAX_THREADS_DIM)
      threads_z = dom[dev].Gfx.knb;
    else
      threads_z = MAX_THREADS_DIM;

    blocks_y = (int)ceil((real) dom[dev].Gfx.jnb / (real) threads_y);
    blocks_z = (int)ceil((real) dom[dev].Gfx.knb / (real) threads_z);

    dim3 dimBlocks_u(threads_y, threads_z);
    dim3 numBlocks_u(blocks_y, blocks_z);

    reset_flag_u<<<numBlocks_u, dimBlocks_u>>>(_flag_u[dev], _dom[dev], bc);

    // reset flag_v
    if(dom[dev].Gfy.kn < MAX_THREADS_DIM)
      threads_z = dom[dev].Gfy.knb;
    else
      threads_z = MAX_THREADS_DIM;

    if(dom[dev].Gfy.in < MAX_THREADS_DIM)
      threads_x = dom[dev].Gfy.inb;
    else
      threads_x = MAX_THREADS_DIM;

    blocks_z = (int)ceil((real) dom[dev].Gfy.knb / (real) threads_z);
    blocks_x = (int)ceil((real) dom[dev].Gfy.inb / (real) threads_x);

    dim3 dimBlocks_v(threads_z, threads_x);
    dim3 numBlocks_v(blocks_z, blocks_x);

    reset_flag_v<<<numBlocks_v, dimBlocks_v>>>(_flag_v[dev], _dom[dev], bc);

    // reset flag_w
    if(dom[dev].Gfz.in < MAX_THREADS_DIM)
      threads_x = dom[dev].Gfz.inb;
    else
      threads_x = MAX_THREADS_DIM;

    if(dom[dev].Gfz.jn < MAX_THREADS_DIM)
      threads_y = dom[dev].Gfz.jnb;
    else
      threads_y = MAX_THREADS_DIM;

    blocks_x = (int)ceil((real) dom[dev].Gfz.inb / (real) threads_x);
    blocks_y = (int)ceil((real) dom[dev].Gfz.jnb / (real) threads_y);

    dim3 dimBlocks_w(threads_x, threads_y);
    dim3 numBlocks_w(blocks_x, blocks_y);

    reset_flag_w<<<numBlocks_w, dimBlocks_w>>>(_flag_w[dev], _dom[dev], bc);


 // push point_particle information to device
    checkCudaErrors(hipMemcpy(_points[dev], points, sizeof(point_struct) * npoints,
      hipMemcpyHostToDevice));



  
  
      // fill in ghost cells for periodic boundary conditions
      if(bc.uW == PERIODIC) {
        blocks_y = (int)ceil((real) dom[dev].Gfx.jnb / (real) threads_y);
        blocks_z = (int)ceil((real) dom[dev].Gfx.knb / (real) threads_z);
        numBlocks_u.x = blocks_y;
        numBlocks_u.y = blocks_z;
        cage_flag_u_periodic_W<<<numBlocks_u, dimBlocks_u>>>(_flag_u[dev],
          _dom[dev]);
      }
      if(bc.uE == PERIODIC) {
        blocks_y = (int)ceil((real) dom[dev].Gfx.jnb / (real) threads_y);
        blocks_z = (int)ceil((real) dom[dev].Gfx.knb / (real) threads_z);
        numBlocks_u.x = blocks_y;
        numBlocks_u.y = blocks_z;
        cage_flag_u_periodic_E<<<numBlocks_u, dimBlocks_u>>>(_flag_u[dev],
          _dom[dev]);
      }
      if(bc.uS == PERIODIC) {
        blocks_z = (int)ceil((real) dom[dev].Gfx.knb / (real) threads_z);
        blocks_x = (int)ceil((real) dom[dev].Gfx.inb / (real) threads_x);
        numBlocks_u.x = blocks_z;
        numBlocks_u.y = blocks_x;
        cage_flag_u_periodic_S<<<numBlocks_u, dimBlocks_u>>>(_flag_u[dev],
          _dom[dev]);
      }
      if(bc.uN == PERIODIC) {
        blocks_z = (int)ceil((real) dom[dev].Gfx.knb / (real) threads_z);
        blocks_x = (int)ceil((real) dom[dev].Gfx.inb / (real) threads_x);
        numBlocks_u.x = blocks_z;
        numBlocks_u.y = blocks_x;
        cage_flag_u_periodic_N<<<numBlocks_u, dimBlocks_u>>>(_flag_u[dev],
          _dom[dev]);
      }
      if(bc.uB == PERIODIC) {
        blocks_x = (int)ceil((real) dom[dev].Gfx.inb / (real) threads_x);
        blocks_y = (int)ceil((real) dom[dev].Gfx.jnb / (real) threads_y);
        numBlocks_u.x = blocks_x;
        numBlocks_u.y = blocks_y;
        cage_flag_u_periodic_B<<<numBlocks_u, dimBlocks_u>>>(_flag_u[dev],
          _dom[dev]);
      }
      if(bc.uT == PERIODIC) {
        blocks_x = (int)ceil((real) dom[dev].Gfx.inb / (real) threads_x);
        blocks_y = (int)ceil((real) dom[dev].Gfx.jnb / (real) threads_y);
        numBlocks_u.x = blocks_x;
        numBlocks_u.y = blocks_y;
        cage_flag_u_periodic_T<<<numBlocks_u, dimBlocks_u>>>(_flag_u[dev],
          _dom[dev]);
      }



      if(bc.vW == PERIODIC) {
        blocks_y = (int)ceil((real) dom[dev].Gfy.jnb / (real) threads_y);
        blocks_z = (int)ceil((real) dom[dev].Gfy.knb / (real) threads_z);
        numBlocks_v.x = blocks_y;
        numBlocks_v.y = blocks_z;
        cage_flag_v_periodic_W<<<numBlocks_v, dimBlocks_v>>>(_flag_v[dev],
          _dom[dev]);
      }
      if(bc.vE == PERIODIC) {
        blocks_y = (int)ceil((real) dom[dev].Gfy.jnb / (real) threads_y);
        blocks_z = (int)ceil((real) dom[dev].Gfy.knb / (real) threads_z);
        numBlocks_v.x = blocks_y;
        numBlocks_v.y = blocks_z;
        cage_flag_v_periodic_E<<<numBlocks_v, dimBlocks_v>>>(_flag_v[dev],
          _dom[dev]);
      }
      if(bc.vS == PERIODIC) {
        blocks_z = (int)ceil((real) dom[dev].Gfy.knb / (real) threads_z);
        blocks_x = (int)ceil((real) dom[dev].Gfy.inb / (real) threads_x);
        numBlocks_v.x = blocks_z;
        numBlocks_v.y = blocks_x;
        cage_flag_v_periodic_S<<<numBlocks_v, dimBlocks_v>>>(_flag_v[dev],
          _dom[dev]);
      }
      if(bc.vN == PERIODIC) {
        blocks_z = (int)ceil((real) dom[dev].Gfy.knb / (real) threads_z);
        blocks_x = (int)ceil((real) dom[dev].Gfy.inb / (real) threads_x);
        numBlocks_v.x = blocks_z;
        numBlocks_v.y = blocks_x;
        cage_flag_v_periodic_N<<<numBlocks_v, dimBlocks_v>>>(_flag_v[dev],
          _dom[dev]);
      }
      if(bc.vB == PERIODIC) {
        blocks_x = (int)ceil((real) dom[dev].Gfy.inb / (real) threads_x);
        blocks_y = (int)ceil((real) dom[dev].Gfy.jnb / (real) threads_y);
        numBlocks_v.x = blocks_x;
        numBlocks_v.y = blocks_y;
        cage_flag_v_periodic_B<<<numBlocks_v, dimBlocks_v>>>(_flag_v[dev],
          _dom[dev]);
      }
      if(bc.vT == PERIODIC) {
        blocks_x = (int)ceil((real) dom[dev].Gfy.inb / (real) threads_x);
        blocks_y = (int)ceil((real) dom[dev].Gfy.jnb / (real) threads_y);
        numBlocks_v.x = blocks_x;
        numBlocks_v.y = blocks_y;
        cage_flag_v_periodic_T<<<numBlocks_v, dimBlocks_v>>>(_flag_v[dev],
          _dom[dev]);
      }




      if(bc.wW == PERIODIC) {
        blocks_y = (int)ceil((real) dom[dev].Gfz.jnb / (real) threads_y);
        blocks_z = (int)ceil((real) dom[dev].Gfz.knb / (real) threads_z);
        numBlocks_w.x = blocks_y;
        numBlocks_w.y = blocks_z;
        cage_flag_w_periodic_W<<<numBlocks_w, dimBlocks_w>>>(_flag_w[dev],
          _dom[dev]);
      }
      if(bc.wE == PERIODIC) {
        blocks_y = (int)ceil((real) dom[dev].Gfz.jnb / (real) threads_y);
        blocks_z = (int)ceil((real) dom[dev].Gfz.knb / (real) threads_z);
        numBlocks_w.x = blocks_y;
        numBlocks_w.y = blocks_z;
        cage_flag_w_periodic_E<<<numBlocks_w, dimBlocks_w>>>(_flag_w[dev],
          _dom[dev]);
      }
      if(bc.wS == PERIODIC) {
        blocks_z = (int)ceil((real) dom[dev].Gfz.knb / (real) threads_z);
        blocks_x = (int)ceil((real) dom[dev].Gfz.inb / (real) threads_x);
        numBlocks_w.x = blocks_z;
        numBlocks_w.y = blocks_x;
        cage_flag_w_periodic_S<<<numBlocks_w, dimBlocks_w>>>(_flag_w[dev],
          _dom[dev]);
      }
      if(bc.wN == PERIODIC) {
        blocks_z = (int)ceil((real) dom[dev].Gfz.knb / (real) threads_z);
        blocks_x = (int)ceil((real) dom[dev].Gfz.inb / (real) threads_x);
        numBlocks_w.x = blocks_z;
        numBlocks_w.y = blocks_x;
        cage_flag_w_periodic_N<<<numBlocks_w, dimBlocks_w>>>(_flag_w[dev],
          _dom[dev]);
      }
      if(bc.wB == PERIODIC) {
        blocks_x = (int)ceil((real) dom[dev].Gfz.inb / (real) threads_x);
        blocks_y = (int)ceil((real) dom[dev].Gfz.jnb / (real) threads_y);
        numBlocks_w.x = blocks_x;
        numBlocks_w.y = blocks_y;
        cage_flag_w_periodic_B<<<numBlocks_w, dimBlocks_w>>>(_flag_w[dev],
          _dom[dev]);
      }
      if(bc.wT == PERIODIC) {
        blocks_x = (int)ceil((real) dom[dev].Gfz.inb / (real) threads_x);
        blocks_y = (int)ceil((real) dom[dev].Gfz.jnb / (real) threads_y);
        numBlocks_w.x = blocks_x;
        numBlocks_w.y = blocks_y;
        cage_flag_w_periodic_T<<<numBlocks_w, dimBlocks_w>>>(_flag_w[dev],
          _dom[dev]);
                           }
   }
  
}


