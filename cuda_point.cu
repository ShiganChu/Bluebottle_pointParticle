#include "hip/hip_runtime.h"
//#include "cuda_point.h"
//#include "point.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

//#include "bluebottle.h"
#include "cuda_point.h"
#include "scalar.h"
#include "entrySearch.h"

extern "C"
real cuda_sum_points_Fz(void)
{
  // results from all devices
//  real *pdt;
 if(npoints<=0) return 0;
  real *Fz; 

  checkCudaErrors(hipMalloc((void**) &(Fz),sizeof(real) * npoints));

  dim3 dimBlocks_p,numBlocks_p;
  block_thread_point(dimBlocks_p,numBlocks_p,npoints);

  points_Fz<<<numBlocks_p,dimBlocks_p>>>(_points[0],Fz,npoints);
    
  real sumFz;
  sumFz = sum_entries(npoints, Fz); 

    
  hipFree(Fz);
 
  return sumFz;
}


//Find the maximum diameter of particles, return the 3*max_dp for poly-disperse particles. Return max_dp for mono-disperse particles.
extern "C"
void cuda_find_DIFF_dt_points(void)
{
  // results from all devices
//  real *pdt;
 if(npoints<=0) return ;
  real *dp; 

  checkCudaErrors(hipMalloc((void**) &(dp),sizeof(real) * npoints));

  dim3 dimBlocks_p,numBlocks_p;
  block_thread_point(dimBlocks_p,numBlocks_p,npoints);

  points_dp<<<numBlocks_p,dimBlocks_p>>>(_points[0],dp,npoints);
    
  real max_dp,min_dp;
  max_dp = find_max(npoints, dp); 
  min_dp = find_min(npoints, dp); 
    
  hipFree(dp);
 
  real obj_dp;

  if((max_dp-min_dp)/max_dp<EPSILON) 
	obj_dp=3*max_dp;
  else 
	obj_dp=max_dp;

real dx=Dom.dx;real dy=Dom.dy;real dz=Dom.dz;
real min_meshsize=min(min(dx,dy),dz);
real max_meshsize=max(max(dx,dy),dz);

DIFF_dt=(obj_dp*obj_dp-min_meshsize*min_meshsize)/16.f/logf(2.f);

/*
printf("\nDIFF_dt %f %f %f\n",DIFF_dt,obj_dp,min_meshsize);
fflush(stdout);
*/
}

//TODO should be moved to the sub-timestep of scalar&&point
extern "C"
real cuda_find_dt_points(real dt)
{
  // results from all devices
//  real *pdt;
 if(npoints<=0) return dt;
  real *vel2_p;

//  checkCudaErrors(hipMalloc((void**) &(pdt),sizeof(real) * npoints));
  checkCudaErrors(hipMalloc((void**) &(vel2_p),sizeof(real) * npoints));

  dim3 dimBlocks_p,numBlocks_p;
  block_thread_point(dimBlocks_p,numBlocks_p,npoints);

  points_vel_square<<<numBlocks_p,dimBlocks_p>>>(_points[0],vel2_p,npoints);

//  copy_points_dt<<<numBlocks_p,dimBlocks_p>>>(pdt,_points[0],npoints);
  real max_vel2;
  max_vel2 = find_max(npoints, vel2_p);
//  min = find_min(npoints, pdt);

  // clean up
//  hipFree(pdt);
  hipFree(vel2_p);

 real pdt;
 real min_meshsize;
 min_meshsize=min(Dom.dx,Dom.dy);
 min_meshsize=min(Dom.dz,min_meshsize);

 pdt=CFL*min_meshsize/(sqrt(max_vel2)+EPSILON);

//printf("\npoints_dt2 %f %f %f\n",pdt,dt,max_vel2);
  if(pdt>dt) pdt=dt; 


  return pdt;
}


/* The base function of the maximum search algorithm. */
int find_max_int(int size, int *d_iarr)
{
  int blocks = 0;
  int threads = 0;

  getNumBlocksAndThreads(size, blocks, threads);

  // create minarr on device
  int h_bytes = blocks * sizeof(int);
  int *d_max_intarr = NULL;
  checkCudaErrors(hipMalloc((void**)&d_max_intarr, h_bytes));
  gpumem += h_bytes;

  hipDeviceSynchronize();

  dim3 dimBlock(threads, 1, 1);
  dim3 dimGrid(blocks, 1, 1);
  int smemSize = threads * sizeof(int);

  // run kernel
  entrySearch_max_int_kernel<<<dimGrid, dimBlock, smemSize>>>(d_iarr,
    d_max_intarr, size);

  getLastCudaError("Kernel execution failed.");
  hipDeviceSynchronize();

  // if there was more than one block, re-run the kernel on the maximum values 
  // from each of the blocks, which now reside in the first block_number indices
  // in d_minarr
  while(blocks > 1) {
    // use only the first block_number indices in min_arr
    size = blocks;
    getNumBlocksAndThreads(size, blocks, threads);

    entrySearch_max_int_kernel<<<dimGrid, dimBlock, smemSize>>>(d_max_intarr,
      d_max_intarr, size);

    getLastCudaError("Kernel execution failed.");
    hipDeviceSynchronize();
  }

  // grab final answer
  int max;
  checkCudaErrors(hipMemcpy(&max, d_max_intarr, sizeof(int),
    hipMemcpyDeviceToHost));
  checkCudaErrors(hipFree(d_max_intarr));
  return max;
}




extern "C"
void cuda_flow_stress()
{
if(npoints<=0) return;
  #pragma omp parallel num_threads(nsubdom)
  {
    int dev = omp_get_thread_num();
    checkCudaErrors(hipSetDevice(dev + dev_start));
//printf("\ndev in cuda_flow_stress %d %d\n",dev,dev_start);
    int threads_x = 0;
    int threads_y = 0;
    int threads_z = 0;
    int blocks_x = 0;
    int blocks_y = 0;
    int blocks_z = 0;

   // u-component
    if(dom[dev].Gfx.jnb < MAX_THREADS_DIM)
      threads_y = dom[dev].Gfx.jnb + 2;
    else
      threads_y = MAX_THREADS_DIM;

    if(dom[dev].Gfx.knb < MAX_THREADS_DIM)
      threads_z = dom[dev].Gfx.knb + 2;
    else
      threads_z = MAX_THREADS_DIM;

    blocks_y = (int)ceil((real) dom[dev].Gfx.jnb / (real) (threads_y-2));
    blocks_z = (int)ceil((real) dom[dev].Gfx.knb / (real) (threads_z-2));



    dim3 dimBlocks_u(threads_y, threads_z);
    dim3 numBlocks_u(blocks_y, blocks_z);




// v-component
    if(dom[dev].Gfy.knb < MAX_THREADS_DIM)
      threads_z = dom[dev].Gfy.knb + 2;
    else
      threads_z = MAX_THREADS_DIM;

    if(dom[dev].Gfy.inb < MAX_THREADS_DIM)
      threads_x = dom[dev].Gfy.inb + 2;
    else
      threads_x = MAX_THREADS_DIM;
  
    blocks_z = (int)ceil((real) dom[dev].Gfy.knb / (real) (threads_z-2));
    blocks_x = (int)ceil((real) dom[dev].Gfy.inb / (real) (threads_x-2));

    dim3 dimBlocks_v(threads_z, threads_x);
    dim3 numBlocks_v(blocks_z, blocks_x);



// w-component
    if(dom[dev].Gfz.inb < MAX_THREADS_DIM)
      threads_x = dom[dev].Gfz.inb + 2;
    else
      threads_x = MAX_THREADS_DIM;

    if(dom[dev].Gfz.jnb < MAX_THREADS_DIM)
      threads_y = dom[dev].Gfz.jnb + 2;
    else
      threads_y = MAX_THREADS_DIM;

    blocks_x = (int)ceil((real) dom[dev].Gfz.inb / (real) (threads_x-2));
    blocks_y = (int)ceil((real) dom[dev].Gfz.jnb / (real) (threads_y-2));

    dim3 dimBlocks_w(threads_x, threads_y);
    dim3 numBlocks_w(blocks_x, blocks_y);

 
    stress_u<<<numBlocks_u, dimBlocks_u>>>(rho_f, nu,_u[dev],_p[dev],_p0[dev], _stress_u[dev], _dom[dev],_flag_u[dev],dt,dt0);
    stress_v<<<numBlocks_v, dimBlocks_v>>>(rho_f, nu,_v[dev],_p[dev],_p0[dev], _stress_v[dev], _dom[dev],_flag_v[dev],dt,dt0);
    stress_w<<<numBlocks_w, dimBlocks_w>>>(rho_f, nu,_w[dev],_p[dev],_p0[dev], _stress_w[dev], _dom[dev],_flag_w[dev],dt,dt0);
 
if(lpt_twoway>0)
{
int coordiSys;
    coordiSys=1;
    DvelDt<<<numBlocks_u, dimBlocks_u>>>(_u0[dev],_u[dev],_conv_u[dev], _dudt[dev], _dom[dev],dt,coordiSys );
    coordiSys=2;
    DvelDt<<<numBlocks_v, dimBlocks_v>>>(_v0[dev],_v[dev],_conv_v[dev], _dvdt[dev], _dom[dev],dt,coordiSys );
    coordiSys=3;
    DvelDt<<<numBlocks_w, dimBlocks_w>>>(_w0[dev],_w[dev],_conv_w[dev], _dwdt[dev], _dom[dev],dt,coordiSys );
}

fflush(stdout);
 }

}

void point_ms_init(void)
{

if(npoints<=0) return;
 // allocate device memory on device
  #pragma omp parallel num_threads(nsubdom)
  {
    int dev = omp_get_thread_num();
    checkCudaErrors(hipSetDevice(dev + dev_start));

    dim3 dimBlocks_p,numBlocks_p;
    block_thread_point(dimBlocks_p,numBlocks_p,npoints);

//bc is bc.uTD etc. Make sure which BC this is. 
	point_ms_initD<<<numBlocks_p, dimBlocks_p>>>(_points[dev],npoints,sc_init_percent);

	}

}

void match_point_vel_with_flow(void)
{

if(npoints<=0) return;
 // allocate device memory on device
  #pragma omp parallel num_threads(nsubdom)
  {
    int dev = omp_get_thread_num();
    checkCudaErrors(hipSetDevice(dev + dev_start));

    dim3 dimBlocks_p,numBlocks_p;
    block_thread_point(dimBlocks_p,numBlocks_p,npoints);

//bc is bc.uTD etc. Make sure which BC this is. 
/*
	point_interp_init<<<numBlocks_p, dimBlocks_p>>>(npoints,_points[dev],
                                                ug[dev],vg[dev],wg[dev],
                                                lpt_stress_u[dev],lpt_stress_v[dev],lpt_stress_w[dev],scg[dev]);
*/
point_interp_init<<<numBlocks_p, dimBlocks_p>>>(npoints,_points[dev],
                                                ug[dev],vg[dev],wg[dev]);
point_interp_init<<<numBlocks_p, dimBlocks_p>>>(npoints,_points[dev],
                                                lpt_stress_u[dev],lpt_stress_v[dev],lpt_stress_w[dev]);
array_init<<<numBlocks_p, dimBlocks_p>>>(scg[dev],_dom[dev],npoints, 0);                    

      interpolate_point_vel_Lag2<<<numBlocks_p, dimBlocks_p>>>(_u[dev],_v[dev],_w[dev],
                                                         npoints,rho_f,nu,
                                                         ug[dev],vg[dev],wg[dev],
                                                        _points[dev],_dom[dev],bc);

     point_vel_specify<<<numBlocks_p, dimBlocks_p>>>(ug[dev],vg[dev],wg[dev],_points[dev],npoints);

 


if(C_stress>0||C_add>0)
interpolate_point_vel_Lag2<<<numBlocks_p, dimBlocks_p>>>(_stress_u[dev],_stress_v[dev],_stress_w[dev],
                                                         npoints, rho_f, nu,
                                                         lpt_stress_u[dev],lpt_stress_v[dev],lpt_stress_w[dev],
                                                         _points[dev],_dom[dev],bc);

	drag_move_points_init<<<numBlocks_p, dimBlocks_p>>>(_points[dev],_dom[dev],npoints,
ug[dev],vg[dev],wg[dev],
lpt_stress_u[dev],lpt_stress_v[dev],lpt_stress_w[dev],
rho_f,mu,g,gradP,
C_add, C_stress,C_drag,
sc_eq,DIFF);

	}

}


void match_bubble_vel_with_flow(void)
{

if(npoints<=0) return;
 // allocate device memory on device
  #pragma omp parallel num_threads(nsubdom)
  {
    int dev = omp_get_thread_num();
    checkCudaErrors(hipSetDevice(dev + dev_start));

    dim3 dimBlocks_p,numBlocks_p;
    block_thread_point(dimBlocks_p,numBlocks_p,npoints);

//bc is bc.uTD etc. Make sure which BC this is. 
//	point_interp_init<<<numBlocks_p, dimBlocks_p>>>(npoints,_points[dev],ug[dev],vg[dev],wg[dev],           lpt_stress_u[dev],lpt_stress_v[dev],lpt_stress_w[dev],scg[dev]);
 
        interpolate_point_vel_Lag2<<<numBlocks_p, dimBlocks_p>>>(_u[dev],_v[dev],_w[dev],
                                                         npoints,rho_f,nu,
                                                         ug[dev],vg[dev],wg[dev],
                                                        _points[dev],_dom[dev],bc);

        point_vel_specify<<<numBlocks_p, dimBlocks_p>>>(ug[dev],vg[dev],wg[dev],_points[dev],npoints);


	store_pointsD<<<numBlocks_p, dimBlocks_p>>>(_points[dev],_dom[dev],npoints);

	}

}



extern "C"
void cuda_move_points()
{


   // parallelize over CPU threads
  #pragma omp parallel num_threads(nsubdom)
  {
    int dev = omp_get_thread_num();
    checkCudaErrors(hipSetDevice(dev + dev_start));
/*
    int threads = MAX_THREADS_1D;
    int blocks = (int)ceil((real) npoints / (real) threads);

    dim3 dimBlocks(threads);
    dim3 numBlocks(blocks);
*/

    dim3 dimBlocks_p,numBlocks_p;
    block_thread_point(dimBlocks_p,numBlocks_p,npoints);

    
    if(npoints > 0) {
//bc is bc.uTD etc. Make sure which BC this is. 
point_interp_init<<<numBlocks_p, dimBlocks_p>>>(npoints,_points[dev],
						ug[dev],vg[dev],wg[dev]);
point_interp_init<<<numBlocks_p, dimBlocks_p>>>(npoints,_points[dev],
						lpt_stress_u[dev],lpt_stress_v[dev],lpt_stress_w[dev]);
array_init<<<numBlocks_p, dimBlocks_p>>>(scg[dev],_dom[dev],npoints, 0);	

if(C_lift>0) 
{
array_init<<<numBlocks_p, dimBlocks_p>>>(lpt_omegaX[dev],_dom[dev],npoints, 0);	
array_init<<<numBlocks_p, dimBlocks_p>>>(lpt_omegaY[dev],_dom[dev],npoints, 0);	
array_init<<<numBlocks_p, dimBlocks_p>>>(lpt_omegaZ[dev],_dom[dev],npoints, 0);	

interpolate_point_scalar_Lag2<<<numBlocks_p, dimBlocks_p>>>(npoints,_omega_x[dev],lpt_omegaX[dev],_points[dev],_dom[dev]);
interpolate_point_scalar_Lag2<<<numBlocks_p, dimBlocks_p>>>(npoints,_omega_y[dev],lpt_omegaY[dev],_points[dev],_dom[dev]);
interpolate_point_scalar_Lag2<<<numBlocks_p, dimBlocks_p>>>(npoints,_omega_z[dev],lpt_omegaZ[dev],_points[dev],_dom[dev]);

}
					
if(lpt_twoway>0)
point_interp_init<<<numBlocks_p, dimBlocks_p>>>(npoints,_points[dev],
						lpt_dudt[dev],lpt_dvdt[dev],lpt_dwdt[dev]);





fflush(stdout);

interpolate_point_vel_Lag2<<<numBlocks_p, dimBlocks_p>>>(_u[dev],_v[dev],_w[dev],
							 npoints,rho_f,nu,
							 ug[dev],vg[dev],wg[dev],
							_points[dev],_dom[dev],bc);
fflush(stdout);

interpolate_point_scalar_Lag2<<<numBlocks_p, dimBlocks_p>>>(npoints,_sc[dev],scg[dev],_points[dev],_dom[dev]);
fflush(stdout);



/*
C_add=0.5;
C_stress=1;
C_drag=1;
*/

//get lpt_stress
//TODO _stress_u is not available near the boundary(set to 0), while lpt_stress can be interpolated on BC
if(C_stress>0||C_add>0) 
{
interpolate_point_vel_Lag2<<<numBlocks_p, dimBlocks_p>>>(_stress_u[dev],_stress_v[dev],_stress_w[dev],
							 npoints, rho_f, nu,
							 lpt_stress_u[dev],lpt_stress_v[dev],lpt_stress_w[dev],
							 _points[dev],_dom[dev],bc);
if(lpt_twoway>0) 
interpolate_point_vel_Lag2<<<numBlocks_p, dimBlocks_p>>>(_dudt[dev],_dvdt[dev],_dwdt[dev],
							 npoints, rho_f, nu,
							 lpt_dudt[dev],lpt_dvdt[dev],lpt_dwdt[dev],
							 _points[dev],_dom[dev],bc);

}

/*
drag_points<<<numBlocks_p, dimBlocks_p>>>(_points[dev],npoints,
ug[dev],vg[dev],wg[dev],
lpt_stress_u[dev],lpt_stress_v[dev],lpt_stress_w[dev],scg[dev],
rho_f,mu,g,gradP,
C_add, C_stress,C_drag,
sc_eq,DIFF);
fflush(stdout);

      move_points_a<<<numBlocks_p, dimBlocks_p>>>(_points[dev], npoints,dt_try);

drag_points<<<numBlocks_p, dimBlocks_p>>>(_points[dev],npoints,
ug[dev],vg[dev],wg[dev],
lpt_stress_u[dev],lpt_stress_v[dev],lpt_stress_w[dev],scg[dev],
rho_f,mu,g,gradP,
C_add, C_stress,C_drag,
sc_eq,DIFF);
fflush(stdout);

      move_points_b<<<numBlocks_p, dimBlocks_p>>>(_dom[dev], _points[dev], npoints,dt_try);
*/
if(lpt_twoway<=0)
{
drag_move_points<<<numBlocks_p, dimBlocks_p>>>(_points[dev],_dom[dev],npoints,
ug[dev],vg[dev],wg[dev],
lpt_stress_u[dev],lpt_stress_v[dev],lpt_stress_w[dev],
lpt_omegaX[dev],lpt_omegaY[dev],lpt_omegaZ[dev],
scg[dev],
rho_f,mu,g,gradP,
C_add, C_stress,C_drag,C_lift,
sc_eq,DIFF,dt_try);
}
else
{
drag_move_points_twoway<<<numBlocks_p, dimBlocks_p>>>(_points[dev],_dom[dev],npoints,
ug[dev],vg[dev],wg[dev],
lpt_stress_u[dev],lpt_stress_v[dev],lpt_stress_w[dev],
lpt_dudt[dev],lpt_dvdt[dev],lpt_dwdt[dev],
lpt_omegaX[dev],lpt_omegaY[dev],lpt_omegaZ[dev],
scg[dev],
rho_f,mu,g,gradP,
C_add, C_stress,C_drag,C_lift,
sc_eq,DIFF,dt_try);
}

/*
//In this subroutine, sc_eq is not used at all
drag_move_bubbles<<<numBlocks_p, dimBlocks_p>>>(_points[dev],_dom[dev],npoints,
ug[dev],vg[dev],wg[dev],
lpt_stress_u[dev],lpt_stress_v[dev],lpt_stress_w[dev],scg[dev],
rho_f,mu,g,gradP,
sc_eq,DIFF,dt_try);
*/

fflush(stdout);

getLastCudaError("Kernel execution failed.");



 /*
      checkCudaErrors(hipFree(forces));
      checkCudaErrors(hipFree(moments));
 */
   		}
  	
    }
}



extern "C"
void lpt_point_twoway_forcing()
{
if(npoints<=0) return;
 // parallelize over CPU threads
  #pragma omp parallel num_threads(nsubdom)
  {
    int dev = omp_get_thread_num();
    checkCudaErrors(hipSetDevice(dev + dev_start));

    dim3 dimBlocks_x,numBlocks_x;
    dim3 dimBlocks_y,numBlocks_y;
    dim3 dimBlocks_z,numBlocks_z;

    int coordiSys,planeDirc;
	
    coordiSys=1;
    planeDirc=1;
    block_thread_cell_noOverLap(dimBlocks_x,numBlocks_x,dom[dev],coordiSys,planeDirc);
    coordiSys=2;
    planeDirc=2;
    block_thread_cell_noOverLap(dimBlocks_y,numBlocks_y,dom[dev],coordiSys,planeDirc);
    coordiSys=3;
    planeDirc=3;
    block_thread_cell_noOverLap(dimBlocks_z,numBlocks_z,dom[dev],coordiSys,planeDirc);
//At the very first step, we add nothing, since particle has no drag on fluid at all
if(dt0 > 0.)
	{
//printf("\nforcing test %f %f\n",dt,dt0);
//TODO figure out why using dt0 rather than dt, besides why there is pike???
//real idt=1/dt0;
real idt=1/dt;
//The coefficient here should be inverse of the present time step rather than last time step
    forcing_add_x_field<<<numBlocks_x, dimBlocks_x>>>(idt, _lpt_mom_source_x[dev],
      _f_x[dev], _dom[dev]);
    forcing_add_y_field<<<numBlocks_y, dimBlocks_y>>>(idt, _lpt_mom_source_y[dev],
      _f_y[dev], _dom[dev]);
    forcing_add_z_field<<<numBlocks_z, dimBlocks_z>>>(idt, _lpt_mom_source_z[dev],
      _f_z[dev], _dom[dev]);

/*
boundary_face_value_homo_end<<<numBlocks_z,dimBlocks_z>>>(_dom[dev],_lpt_mom_source_z[dev],3);
real mom=sum_entries(Dom.Gfz.s3b,_lpt_mom_source_z[dev]);
//printf("\nlpt_mom_test %f %f %f %f\n",mom,dt0,dt,mom/dt0);
real Vcell=powf(Dom.dx,3);
printf("\nlpt_mom_test %f %f\n",mom*Vcell,mom/dt0);
fflush(stdout);

 checkCudaErrors(hipMemcpy(_lpt_mom_source_z[dev],_f_z[dev], sizeof(real) * Dom.Gfz.s3b,hipMemcpyDeviceToDevice));
boundary_face_value_homo_end<<<numBlocks_z,dimBlocks_z>>>(_dom[dev],_lpt_mom_source_z[dev],3);
real fz=sum_entries(Dom.Gfz.s3b,_lpt_mom_source_z[dev]);
printf("\nfzTotal %f\n",fz*Vcell);
fflush(stdout);
*/
	}



     forcing_reset_x<<<numBlocks_x, dimBlocks_x>>>(_lpt_mom_source_x[dev], _dom[dev]);
     forcing_reset_y<<<numBlocks_y, dimBlocks_y>>>(_lpt_mom_source_y[dev], _dom[dev]);
     forcing_reset_z<<<numBlocks_z, dimBlocks_z>>>(_lpt_mom_source_z[dev], _dom[dev]);

     }
}


   
extern "C"
void lpt_point_twoway_momentum()
{
if(npoints<=0) return;
 // parallelize over CPU threads
  #pragma omp parallel num_threads(nsubdom)
  {
    int dev = omp_get_thread_num();
    checkCudaErrors(hipSetDevice(dev + dev_start));

int valType=0;

//Mollify particle back reaction momentum source
/*
lpt_mollify_delta_scH(1,valType,dev,_lpt_mom_source_x[dev]);
lpt_mollify_delta_scH(2,valType,dev,_lpt_mom_source_y[dev]);
lpt_mollify_delta_scH(3,valType,dev,_lpt_mom_source_z[dev]);
*/

//Using Gaussian Kernel
lpt_mollify_sc_optH(1,valType,dev,_lpt_mom_source_x[dev]);
lpt_mollify_sc_optH(2,valType,dev,_lpt_mom_source_y[dev]);
lpt_mollify_sc_optH(3,valType,dev,_lpt_mom_source_z[dev]);


     }
}














//extern "C"
void sortParticles(int *dgridParticleHash, int *dgridParticleIndex, int numParticles)
{
//sort dgridParticleIndex based on dgridParticleHash, and they both change order
        thrust::sort_by_key(thrust::device_ptr<int>(dgridParticleHash),
                            thrust::device_ptr<int>(dgridParticleHash + numParticles),
                            thrust::device_ptr<int>(dgridParticleIndex));
}

extern "C"
void lpt_mollify_delta_scH(int coordiSys,int valType,int dev,real *scSrc)
{
if(npoints<=0) return;

//    dim3 dimBlocks_3d,numBlocks_3d;
    dim3 dimBlocks_w,numBlocks_w;
    dim3 dimBlocks_z,numBlocks_z;
    dim3 dimBlocks_p,numBlocks_p;
//    dim3 numBlocks_st;

//    dim3 numBlocks_print,dimBlocks_print;

//int coordiSys=0;//coordinate systerm, cell-center or face center
int planeDirc=3;//parallel x-y or y-z or x-z plane

//block_thread_cell(dimBlocks_z,numBlocks_z,dom[dev],coordiSys,planeDirc);
block_thread_cell_noOverLap(dimBlocks_w,numBlocks_w,dom[dev],coordiSys,planeDirc);
//incGhost=1;block_thread_cell_3D(dimBlocks_3d,numBlocks_3d,dom[dev],coordiSys,incGhost);
block_thread_point(dimBlocks_p,numBlocks_p,npoints);
//block_thread_point(dimBlocks_p,numBlocks_st,npoints*STENCIL3);


int lenCell;
switch(coordiSys)
{
case 0:lenCell=dom[dev].Gcc.s3b;break;
case 1:lenCell=dom[dev].Gfx.s3b;break;
case 2:lenCell=dom[dev].Gfy.s3b;break;
case 3:lenCell=dom[dev].Gfz.s3b;break;
default: break;
}

    real *scSrc_buf;
    checkCudaErrors(hipMalloc((void**) &(scSrc_buf),sizeof(real) * lenCell));
    scSrc_value_init<<<numBlocks_w,dimBlocks_w>>>(_dom[dev],scSrc_buf,0.f,coordiSys);

//block_thread_point(dimBlocks_print,numBlocks_print,lenCell);

checkCudaErrors(hipMemset(gridParticleHash[dev],-1,npoints*sizeof(int)));
checkCudaErrors(hipMemset(gridParticleIndex[dev],-1,npoints*sizeof(int)));

checkCudaErrors(hipMemset(cellStart[dev],-1,lenCell*sizeof(int)));
checkCudaErrors(hipMemset(cellEnd[dev],-1,lenCell*sizeof(int)));


/*
hipEvent_t start, stop;
hipEventCreate(&start);
hipEventCreate(&stop);
float milliseconds = 0;
*/
//printf("\ngridDim_p %d %d %d\n",numBlocks_p.x,dimBlocks_p.x,npoints);
//printf("\ngridDim_3d %d %d %d %d\n",numBlocks_3d.x,numBlocks_3d.y,numBlocks_3d.z,lenCell);

lpt_delta_point_position<<<numBlocks_p,dimBlocks_p>>>(_points[dev],_dom[dev],
posXold[dev],posYold[dev],posZold[dev],
lptSourceValOld[dev],npoints,coordiSys,valType);
 

calcHash_optD<<<numBlocks_p,dimBlocks_p>>>(gridParticleHash[dev],
					gridParticleIndex[dev],
					_dom[dev],
					posXold[dev],posYold[dev],posZold[dev],
					npoints,coordiSys);


sortParticles(gridParticleHash[dev],gridParticleIndex[dev],npoints);

/*
milliseconds = 0;
hipEventRecord(start);
*/
findCellStart_deltaD<<<numBlocks_p,dimBlocks_p>>>(cellStart[dev],
						cellEnd[dev],
						gridParticleHash[dev],
						gridParticleIndex[dev],
						lptSourceVal[dev],
						lptSourceValOld[dev],
						npoints);
/*
hipEventRecord(stop);
hipEventSynchronize(stop);
hipEventElapsedTime(&milliseconds, start, stop);
printf("\ntime_reoder %f\n",milliseconds);
fflush(stdout);
milliseconds = 0;
hipEventRecord(start);

fflush(stdout);

hipEventRecord(stop);
hipEventSynchronize(stop);
hipEventElapsedTime(&milliseconds, start, stop);
printf("\ntime_weight %f\n",milliseconds);
fflush(stdout);

milliseconds = 0;
hipEventRecord(start);
*/


//Note the numBlocks_w doesn't contain overLap for the shared memory. This should vary from kernel to kernel
lpt_mollify_delta_scD<<<numBlocks_w,dimBlocks_w>>>(_dom[dev],scSrc_buf,lptSourceVal[dev],cellStart[dev],cellEnd[dev],npoints,coordiSys);

if(coordiSys>0)
{
boundary_face_value_periodic_start<<<numBlocks_w,dimBlocks_w>>>(_dom[dev],scSrc_buf,coordiSys);
boundary_face_value_periodic_end<<<numBlocks_w,dimBlocks_w>>>(_dom[dev],scSrc_buf,coordiSys);
}

cuda_scSrc_BC(coordiSys,SCALAR_TYPE,scSrc_buf,dev);

scSrc_value_add<<<numBlocks_w,dimBlocks_w>>>(_dom[dev],scSrc,scSrc_buf,coordiSys);

getLastCudaError("Kernel execution failed.");

    checkCudaErrors(hipFree(scSrc_buf));
/*
fflush(stdout);
hipEventRecord(stop);
hipEventSynchronize(stop);
hipEventElapsedTime(&milliseconds, start, stop);
printf("\ntime_mollify %f\n",milliseconds);
fflush(stdout);
*/
//print_kernel_array_int<<<numBlocks_print,dimBlocks_print>>>(cellEnd[dev],lenCell);
}


extern "C"
void lpt_mollify_sc_optH(int coordiSys,int valType,int dev,real *scSrc)
{
if(npoints<=0) return;

    dim3 dimBlocks_3d,numBlocks_3d;
    dim3 dimBlocks_w,numBlocks_w;
    dim3 dimBlocks_z,numBlocks_z;
    dim3 dimBlocks_p,numBlocks_p;
//    dim3 numBlocks_st;

//    dim3 numBlocks_print,dimBlocks_print;

//int coordiSys=0;//coordinate systerm, cell-center or face center
int planeDirc=3;//parallel x-y or y-z or x-z plane
int incGhost=0;
block_thread_cell(dimBlocks_w,numBlocks_w,dom[dev],coordiSys,planeDirc);
block_thread_cell_noOverLap(dimBlocks_z,numBlocks_z,dom[dev],coordiSys,planeDirc);
block_thread_cell_3D(dimBlocks_3d,numBlocks_3d,dom[dev],coordiSys,incGhost);
block_thread_point(dimBlocks_p,numBlocks_p,npoints);
//block_thread_point(dimBlocks_p,numBlocks_st,npoints*STENCIL3);


int lenCell;
switch(coordiSys)
{
case 0:lenCell=dom[dev].Gcc.s3b;break;
case 1:lenCell=dom[dev].Gfx.s3b;break;
case 2:lenCell=dom[dev].Gfy.s3b;break;
case 3:lenCell=dom[dev].Gfz.s3b;break;
default: break;
}

    real *scSrc_buf;
    checkCudaErrors(hipMalloc((void**) &(scSrc_buf),sizeof(real) * lenCell));
    scSrc_value_init<<<numBlocks_z,dimBlocks_z>>>(_dom[dev],scSrc_buf,0.f,coordiSys);

//block_thread_point(dimBlocks_print,numBlocks_print,lenCell);
getLastCudaError("Kernel execution failed.");

checkCudaErrors(hipMemset(gridParticleHash[dev],-1,npoints*sizeof(int)));
checkCudaErrors(hipMemset(gridParticleIndex[dev],-1,npoints*sizeof(int)));

checkCudaErrors(hipMemset(cellStart[dev],-1,lenCell*sizeof(int)));
checkCudaErrors(hipMemset(cellEnd[dev],-1,lenCell*sizeof(int)));
checkCudaErrors(hipMemset(pointNumInCell[dev],-1,lenCell*sizeof(int)));
checkCudaErrors(hipMemset(gridFlowHash[dev],-1,lenCell*sizeof(int)));

/*
hipEvent_t start, stop;
hipEventCreate(&start);
hipEventCreate(&stop);
float milliseconds = 0;
*/
getLastCudaError("Kernel execution failed.");


//printf("\ngridDim_p %d %d %d\n",numBlocks_p.x,dimBlocks_p.x,npoints);
//fflush(stdout);
//printf("\nnumBlocks_3d %d %d %d %d %d\n",numBlocks_3d.x,numBlocks_3d.y,numBlocks_3d.z,lenCell,Dom.Gcc.in);
//printf("\ndimBlocks_3d %d %d %d\n",dimBlocks_3d.x,dimBlocks_3d.y,dimBlocks_3d.z);

//array_init<<<numBlocks_st, dimBlocks_p>>>(Ksi[dev],_dom[dev],npoints*STENCIL3, 0.);

lpt_point_position<<<numBlocks_p,dimBlocks_p>>>(_points[dev],posXold[dev],posYold[dev],posZold[dev],npoints);

calcHash_optD<<<numBlocks_p,dimBlocks_p>>>(gridParticleHash[dev],
					gridParticleIndex[dev],
					_dom[dev],
					posXold[dev],posYold[dev],posZold[dev],
					npoints,coordiSys);

sortParticles(gridParticleHash[dev],gridParticleIndex[dev],npoints);

findCellStart_optD<<<numBlocks_p,dimBlocks_p>>>(cellStart[dev],
						cellEnd[dev],
						gridParticleHash[dev],
						gridParticleIndex[dev],
						posX[dev],posY[dev],posZ[dev],
						posXold[dev],posYold[dev],posZold[dev],
						npoints);



//valType=1 for particle volume fraction; valType =0 for other cell-centerred parameter 
lpt_point_ksi_opt<<<numBlocks_p,dimBlocks_p>>>(_points[dev],_dom[dev],posX[dev],posY[dev],posZ[dev],Ksi[dev],gridParticleIndex[dev],npoints,coordiSys,valType);
fflush(stdout);

getLastCudaError("Kernel execution failed.");


//Store the grid hash index
calcGridFlowHash_optD<<<numBlocks_z,dimBlocks_z>>>(_dom[dev],gridFlowHash[dev],coordiSys);


//Find the maximum number of particles inside grid cell
calcMaxPointsPerCell_optD<<<numBlocks_z,dimBlocks_z>>>(_dom[dev],cellStart[dev],cellEnd[dev],pointNumInCell[dev],coordiSys);
int maxPointsPerCell=find_max_int(lenCell,pointNumInCell[dev]);


//Gaussian mollfication
lpt_mollify_sc_ksi_optD<<<numBlocks_w,dimBlocks_w>>>(_dom[dev],scSrc_buf,
posX[dev],posY[dev],posZ[dev],
Ksi[dev],cellStart[dev],cellEnd[dev],gridFlowHash[dev],
gridParticleIndex[dev],npoints,maxPointsPerCell,
coordiSys,valType);

getLastCudaError("Kernel execution failed.");

if(coordiSys>0)
{
for(int start_end=1;start_end<=2;start_end++)
	{
	gaussian_periodic_value_add_supplemental<<<numBlocks_z,dimBlocks_z>>>(_dom[dev],scSrc_buf,
	      Ksi[dev],cellStart[dev],cellEnd[dev],gridFlowHash[dev],
              maxPointsPerCell,coordiSys, start_end);
	}
}

/*
if(coordiSys>0)
{
boundary_face_value_periodic_start<<<numBlocks_z,dimBlocks_z>>>(_dom[dev],scSrc_buf,coordiSys);
boundary_face_value_periodic_end<<<numBlocks_z,dimBlocks_z>>>(_dom[dev],scSrc_buf,coordiSys);
}
*/

cuda_scSrc_BC(coordiSys,SCALAR_TYPE, scSrc_buf,dev);

//Difuse the buf value based on Cappeccelo&Desjadins(2012)
cuda_diffScalar_sub_explicitH(coordiSys,dev,scSrc_buf);

////Diffuse the scalar after gaussian mollification
////Explicit solver Takes 4 times longer than one time implicit solver
//////TODO Need to change Gcc to all coordiSys
////cuda_diffScalar_helmholtz_CN(coordiSys,dev,scSrc);

//Add the buf value to the source
scSrc_value_add<<<numBlocks_z,dimBlocks_z>>>(_dom[dev],scSrc,scSrc_buf,coordiSys);

    checkCudaErrors(hipFree(scSrc_buf));

}


//About Swap, and reference, dirc is the system direction, dirc2 is the plane direction
void block_thread_cell(dim3 &dimBlocks,dim3 &numBlocks,dom_struct dom,int dirc,int dirc2)
{

    int threads_y = 0;
    int threads_x = 0;
    int blocks_y = 0;
    int blocks_x = 0;

    int lenX=0;
    int lenY=0;

grid_info G;
switch(dirc)
{
case 0:G=dom.Gcc;break;
case 1:G=dom.Gfx;break;
case 2:G=dom.Gfy;break;
case 3:G=dom.Gfz;break;
default: break;
}

	switch(dirc2)
	{
	case 0:
		lenX=G._inb;
		lenY=G._jnb;
		break;
	case 1:
		lenX=G._jnb;
		lenY=G._knb;
		break;
	case 2:
		lenX=G._knb;
		lenY=G._inb;
		break;
	case 3:
		lenX=G._inb;
		lenY=G._jnb;
		break;
	default: break;	
	}


    if(lenX < MAX_THREADS_DIM)
      threads_x = lenX+2;
    else
      threads_x = MAX_THREADS_DIM;

    if(lenY < MAX_THREADS_DIM)
      threads_y = lenY+2;
    else
      threads_y = MAX_THREADS_DIM;


    blocks_x = (int)ceil((real) lenX / (real) (threads_x-2));
    blocks_y = (int)ceil((real) lenY / (real) (threads_y-2));

    dimBlocks.x=threads_x;
    dimBlocks.y=threads_y;
    numBlocks.x=blocks_x;
    numBlocks.y=blocks_y;

}


//About Swap, and reference, dirc is the system direction, dirc2 is the plane direction
void block_thread_cell_noOverLap(dim3 &dimBlocks,dim3 &numBlocks,dom_struct dom,int dirc,int dirc2)
{

    int threads_y = 0;
    int threads_x = 0;
    int blocks_y = 0;
    int blocks_x = 0;

    int lenX=0;
    int lenY=0;

grid_info G;
switch(dirc)
{
case 0:G=dom.Gcc;break;
case 1:G=dom.Gfx;break;
case 2:G=dom.Gfy;break;
case 3:G=dom.Gfz;break;
default: break;
}

	switch(dirc2)
	{
	case 1:
		lenX=G._jnb;
		lenY=G._knb;
		break;
	case 2:
		lenX=G._knb;
		lenY=G._inb;
		break;
	case 3:
		lenX=G._inb;
		lenY=G._jnb;
		break;
	default: break;	
	}


    if(lenX < MAX_THREADS_DIM)
      threads_x = lenX;
    else
      threads_x = MAX_THREADS_DIM;

    if(lenY < MAX_THREADS_DIM)
      threads_y = lenY;
    else
      threads_y = MAX_THREADS_DIM;


    blocks_x = (int)ceil((real) lenX / (real) (threads_x));
    blocks_y = (int)ceil((real) lenY / (real) (threads_y));

    dimBlocks.x=threads_x;
    dimBlocks.y=threads_y;
    numBlocks.x=blocks_x;
    numBlocks.y=blocks_y;

}


/*
//About Swap, and reference, dirc is the system direction, get 3d blocks and threads
void block_thread_cell_3D(dim3 &dimBlocks,dim3 &numBlocks,dom_struct dom,int dirc)
{

    int threads_x = 0;
    int threads_y = 0;
    int threads_z = 0;
    int blocks_x = 0;
    int blocks_y = 0;
    int blocks_z = 0;

    int lenX=0;
    int lenY=0;
    int lenZ=0;

grid_info G;
switch(dirc)
{
case 0:G=dom.Gcc;break;
case 1:G=dom.Gfx;break;
case 2:G=dom.Gfy;break;
case 3:G=dom.Gfz;break;
default: break;
}

		lenX=G._inb;
		lenY=G._jnb;
		lenZ=G._knb;


    if(lenX < MAX_THREADS_DIM)
      threads_x = lenX;
    else
      threads_x = MAX_THREADS_DIM;

    if(lenY < MAX_THREADS_DIM)
      threads_y = lenY;
    else
      threads_y = MAX_THREADS_DIM;

    int MAX_THREADS_Z=(int) 1024/(1.0f*MAX_THREADS_DIM*MAX_THREADS_DIM);
    if(lenZ < MAX_THREADS_Z)
      threads_z = lenZ;
    else
      threads_z = MAX_THREADS_Z;

    blocks_x = (int)ceil((real) lenX / (real) (threads_x));
    blocks_y = (int)ceil((real) lenY / (real) (threads_y));
    blocks_z = (int)ceil((real) lenZ / (real) (threads_z));

    dimBlocks.x=threads_x;
    dimBlocks.y=threads_y;
    dimBlocks.z=threads_z;

    numBlocks.x=blocks_x;
    numBlocks.y=blocks_y;
    numBlocks.z=blocks_z;
}

*/


//About Swap, and reference, dirc is the system direction, get 3d blocks and threads
//This method is faster than blockDim=16*16*4 by 10%
void block_thread_cell_3D(dim3 &dimBlocks,dim3 &numBlocks,dom_struct dom,int dirc,int incGhost)
{

    int threads_x = 0;
    int threads_y = 0;
    int threads_z = 0;
    int blocks_x = 0;
    int blocks_y = 0;
    int blocks_z = 0;

    int lenX=0;
    int lenY=0;
    int lenZ=0;

grid_info G;
switch(dirc)
{
case 0:G=dom.Gcc;break;
case 1:G=dom.Gfx;break;
case 2:G=dom.Gfy;break;
case 3:G=dom.Gfz;break;
default: break;
}


if(incGhost==1)
{
		lenX=G._inb;
		lenY=G._jnb;
		lenZ=G._knb;
}
else
{
		lenX=G._in;
                lenY=G._jn;
                lenZ=G._kn;
}

    if(lenX < MAX_THREADS_DIM3)
      threads_x = lenX;
    else
      threads_x = MAX_THREADS_DIM3;

    if(lenY < MAX_THREADS_DIM3)
      threads_y = lenY;
    else
      threads_y = MAX_THREADS_DIM3;


int maxThread_z=floor(MAX_THREADS_BLOCK/(1.f*threads_y)/(1.f*threads_x));
    if(lenZ < maxThread_z)
      threads_z = lenZ;
    else
      threads_z = maxThread_z;


/*
    if(lenZ < MAX_THREADS_DIM3)
      threads_z = lenZ;
    else
      threads_z = MAX_THREADS_DIM3;
*/
    blocks_x = (int)ceil((real) lenX / (real) (threads_x));
    blocks_y = (int)ceil((real) lenY / (real) (threads_y));
    blocks_z = (int)ceil((real) lenZ / (real) (threads_z));

    dimBlocks.x=threads_x;
    dimBlocks.y=threads_y;
    dimBlocks.z=threads_z;

    numBlocks.x=blocks_x;
    numBlocks.y=blocks_y;
    numBlocks.z=blocks_z;
}

void block_thread_point(dim3 &dimBlocks,dim3 &numBlocks,int npoints)
{
    int threads = MAX_THREADS_1D;
    int blocks = (int)ceil((real) npoints / (real) threads);

    dimBlocks.x=threads;
    numBlocks.x=blocks;

}


void cuda_malloc_array_int(int** &A,int lenArray)
{
A= (int**) malloc(nsubdom * sizeof(int*));
          cpumem += nsubdom * sizeof(int*);

  #pragma omp parallel num_threads(nsubdom)
  {
    int dev = omp_get_thread_num();
    checkCudaErrors(hipSetDevice(dev + dev_start));

   checkCudaErrors(hipMalloc((void**) &(A[dev]), sizeof(int) * lenArray));
    gpumem += sizeof(int) * lenArray;
  }
}

void cuda_malloc_array_real(real**& A,int lenArray)
{   
A= (real**) malloc(nsubdom * sizeof(real*));
          cpumem += nsubdom * sizeof(real*);
      
  #pragma omp parallel num_threads(nsubdom)
  {
    int dev = omp_get_thread_num();
    checkCudaErrors(hipSetDevice(dev + dev_start));
    

   checkCudaErrors(hipMalloc((void**) &(A[dev]), sizeof(real)*lenArray));
    gpumem += sizeof(real) * lenArray;
  } 
}   


void cuda_free_array_real(real**& A)
{
  // free device memory on device
  #pragma omp parallel num_threads(nsubdom)
  {
    int dev = omp_get_thread_num();
    checkCudaErrors(hipSetDevice(dev + dev_start));
    checkCudaErrors(hipFree(A[dev]));
}

  free(A);

}

void cuda_free_array_int(int** &A)
{
  // free device memory on device
  #pragma omp parallel num_threads(nsubdom)
  {
    int dev = omp_get_thread_num();
    checkCudaErrors(hipSetDevice(dev + dev_start));

    checkCudaErrors(hipFree(A[dev]));
}   
    
  free(A);

}

extern "C"
void cuda_point_malloc(void)
{

  // allocate device memory on host
  _points = (point_struct**) malloc(nsubdom * sizeof(point_struct*));
  cpumem += nsubdom * sizeof(point_struct*);

  _flag_u = (int**) malloc(nsubdom * sizeof(int*));
  cpumem += nsubdom * sizeof(int*);
  _flag_v = (int**) malloc(nsubdom * sizeof(int*));
  cpumem += nsubdom * sizeof(int*);
  _flag_w = (int**) malloc(nsubdom * sizeof(int*));
  cpumem += nsubdom * sizeof(int*);


  // allocate device memory on host
  //add by shigan_9_22_2014, fluid stress on face center
  _stress_u = (real**) malloc(nsubdom * sizeof(real*));
  cpumem += nsubdom * sizeof(real*);
  _stress_v = (real**) malloc(nsubdom * sizeof(real*));
  cpumem += nsubdom * sizeof(real*);
  _stress_w = (real**) malloc(nsubdom * sizeof(real*));
  cpumem += nsubdom * sizeof(real*);


  _lpt_mom_source_x = (real**) malloc(nsubdom * sizeof(real*));
  cpumem += nsubdom * sizeof(real*);
  _lpt_mom_source_y = (real**) malloc(nsubdom * sizeof(real*));
  cpumem += nsubdom * sizeof(real*);
  _lpt_mom_source_z = (real**) malloc(nsubdom * sizeof(real*));
  cpumem += nsubdom * sizeof(real*);




if(lpt_twoway>0)
{
  _dudt = (real**) malloc(nsubdom * sizeof(real*));
  cpumem += nsubdom * sizeof(real*);
  _dvdt = (real**) malloc(nsubdom * sizeof(real*));
  cpumem += nsubdom * sizeof(real*);
  _dwdt = (real**) malloc(nsubdom * sizeof(real*));
  cpumem += nsubdom * sizeof(real*);

}


  lpt_point_source_mollify_init();


  // allocate device memory on device
  #pragma omp parallel num_threads(nsubdom)
  {
    int dev = omp_get_thread_num();
    checkCudaErrors(hipSetDevice(dev + dev_start));

    checkCudaErrors(hipMalloc((void**) &(_points[dev]),
      sizeof(point_struct) * npoints));
    gpumem += sizeof(point_struct) * npoints;

   checkCudaErrors(hipMalloc((void**) &(_flag_u[dev]),
      sizeof(int) * dom[dev].Gfx.s3b));
    gpumem += sizeof(int) * dom[dev].Gfx.s3b;
    checkCudaErrors(hipMalloc((void**) &(_flag_v[dev]),
      sizeof(int) * dom[dev].Gfy.s3b));
    gpumem += sizeof(int) * dom[dev].Gfy.s3b;
    checkCudaErrors(hipMalloc((void**) &(_flag_w[dev]),
      sizeof(int) * dom[dev].Gfz.s3b));
    gpumem += sizeof(int) * dom[dev].Gfz.s3b;
 
//add by shigan
    checkCudaErrors(hipMalloc((void**) &(_stress_u[dev]),
      sizeof(real) * dom[dev].Gfx.s3b));
    gpumem += sizeof(int) * dom[dev].Gfx.s3b;
    checkCudaErrors(hipMalloc((void**) &(_stress_v[dev]),
      sizeof(real) * dom[dev].Gfy.s3b));
    gpumem += sizeof(int) * dom[dev].Gfy.s3b;
    checkCudaErrors(hipMalloc((void**) &(_stress_w[dev]),
      sizeof(real) * dom[dev].Gfz.s3b));
    gpumem += sizeof(int) * dom[dev].Gfz.s3b;

if(lpt_twoway>0)
{
    checkCudaErrors(hipMalloc((void**) &(_dudt[dev]),
      sizeof(real) * dom[dev].Gfx.s3b));
    gpumem += sizeof(int) * dom[dev].Gfx.s3b;
    checkCudaErrors(hipMalloc((void**) &(_dvdt[dev]),
      sizeof(real) * dom[dev].Gfy.s3b));
    gpumem += sizeof(int) * dom[dev].Gfy.s3b;
    checkCudaErrors(hipMalloc((void**) &(_dwdt[dev]),
      sizeof(real) * dom[dev].Gfz.s3b));
    gpumem += sizeof(int) * dom[dev].Gfz.s3b;
}


    checkCudaErrors(hipMalloc((void**) &(_lpt_mom_source_x[dev]),
      sizeof(real) * dom[dev].Gfx.s3b));
    gpumem += sizeof(int) * dom[dev].Gfx.s3b;
    checkCudaErrors(hipMalloc((void**) &(_lpt_mom_source_y[dev]),
      sizeof(real) * dom[dev].Gfy.s3b));
    gpumem += sizeof(int) * dom[dev].Gfy.s3b;
    checkCudaErrors(hipMalloc((void**) &(_lpt_mom_source_z[dev]),
      sizeof(real) * dom[dev].Gfz.s3b));
    gpumem += sizeof(int) * dom[dev].Gfz.s3b;


/*
    checkCudaErrors(hipMalloc((void**) &(_omega_x[dev]),
      sizeof(real) * dom[dev].Gfx.s3b));
    checkCudaErrors(hipMalloc((void**) &(_omega_y[dev]),
      sizeof(real) * dom[dev].Gfy.s3b));
    checkCudaErrors(hipMalloc((void**) &(_omega_z[dev]),
      sizeof(real) * dom[dev].Gfz.s3b));
*/





  }




}

extern "C"
void cuda_point_free(void)
{
  // free device memory on device
  #pragma omp parallel num_threads(nsubdom)
  {
    int dev = omp_get_thread_num();
    checkCudaErrors(hipSetDevice(dev + dev_start));

    checkCudaErrors(hipFree(_points[dev]));
    checkCudaErrors(hipFree(_flag_u[dev]));
    checkCudaErrors(hipFree(_flag_v[dev]));
    checkCudaErrors(hipFree(_flag_w[dev]));
  

    checkCudaErrors(hipFree(_stress_u[dev]));
    checkCudaErrors(hipFree(_stress_v[dev]));
    checkCudaErrors(hipFree(_stress_w[dev]));

if(lpt_twoway>0)
{
    checkCudaErrors(hipFree(_dudt[dev]));
    checkCudaErrors(hipFree(_dvdt[dev]));
    checkCudaErrors(hipFree(_dwdt[dev]));
}

    checkCudaErrors(hipFree(_lpt_mom_source_x[dev]));
    checkCudaErrors(hipFree(_lpt_mom_source_y[dev]));
    checkCudaErrors(hipFree(_lpt_mom_source_z[dev]));

/*
    checkCudaErrors(hipFree(_omega_x[dev]));
    checkCudaErrors(hipFree(_omega_y[dev]));
    checkCudaErrors(hipFree(_omega_z[dev]));
*/


}

 
  free(_points);
  free(_flag_u);
  free(_flag_v);
  free(_flag_w);

  free(_stress_u);
  free(_stress_v);
  free(_stress_w);

if(lpt_twoway>0)
{
  free(_dudt);
  free(_dvdt);
  free(_dwdt);

}

  free(_lpt_mom_source_x);
  free(_lpt_mom_source_y);
  free(_lpt_mom_source_z);

/*  
  free(_omega_x);
  free(_omega_y);
  free(_omega_z);
*/

  lpt_point_source_mollify_final();

}

extern "C"
void cuda_point_push(void)
{
if(npoints<=0) return;
  // copy host data to device
  #pragma omp parallel num_threads(nsubdom)
  {
    int dev = omp_get_thread_num();
    checkCudaErrors(hipSetDevice(dev + dev_start));

    checkCudaErrors(hipMemcpy(_points[dev], points, sizeof(point_struct) * npoints,hipMemcpyHostToDevice));
 }
}

extern "C"
void cuda_point_pull(void)
{
if(npoints<=0) return;
  // all devices have the same point_particle data for now, so just copy one of them
  checkCudaErrors(hipMemcpy(points, _points[0], sizeof(point_struct) * npoints,
    hipMemcpyDeviceToHost));
}



//extern "C"
void lpt_point_source_mollify_init()
{
cuda_malloc_array_real(ug,npoints);
cuda_malloc_array_real(vg,npoints);
cuda_malloc_array_real(wg,npoints);

cuda_malloc_array_real(posX,npoints);
cuda_malloc_array_real(posY,npoints);
cuda_malloc_array_real(posZ,npoints);
cuda_malloc_array_real(posXold,npoints);
cuda_malloc_array_real(posYold,npoints);
cuda_malloc_array_real(posZold,npoints);

cuda_malloc_array_real(lptSourceVal,npoints);
cuda_malloc_array_real(lptSourceValOld,npoints);

cuda_malloc_array_real(lpt_stress_u,npoints);
cuda_malloc_array_real(lpt_stress_v,npoints);
cuda_malloc_array_real(lpt_stress_w,npoints);

cuda_malloc_array_real(lpt_omegaX,npoints);
cuda_malloc_array_real(lpt_omegaY,npoints);
cuda_malloc_array_real(lpt_omegaZ,npoints);


if(lpt_twoway>0)
{
cuda_malloc_array_real(lpt_dudt,npoints);
cuda_malloc_array_real(lpt_dvdt,npoints);
cuda_malloc_array_real(lpt_dwdt,npoints);
}

cuda_malloc_array_real(scg,npoints);
//cuda_malloc_array_real(Weight,npoints);
cuda_malloc_array_real(Ksi,npoints*STENCIL3);

cuda_malloc_array_int(gridParticleIndex,npoints);
cuda_malloc_array_int(gridParticleHash,npoints);

//calculate the maximum length of coordinate system
int lenCell=Dom.Gcc.s3b;
int len1=Dom.Gfx.s3b;
int len2=Dom.Gfy.s3b;
int len3=Dom.Gfz.s3b;
if(lenCell<len1) lenCell=len1;
if(len2<len3) len2=len3;
if(lenCell<len2) lenCell=len2;
//lenCell=max(max(len1,len2),max(len3,lenCell));
cuda_malloc_array_int(cellStart,lenCell);
cuda_malloc_array_int(cellEnd,lenCell);
cuda_malloc_array_int(pointNumInCell,lenCell);
cuda_malloc_array_int(gridFlowHash,lenCell);


  // allocate device memory on device
  #pragma omp parallel num_threads(nsubdom)
  {
    int dev = omp_get_thread_num();
    checkCudaErrors(hipSetDevice(dev + dev_start));

    //Initialize those two arrays since they will be initialized with smaller length later in lpt_mollify_scH
    checkCudaErrors(hipMemset(cellStart[dev],-1,lenCell*sizeof(int)));
    checkCudaErrors(hipMemset(cellEnd[dev],-1,lenCell*sizeof(int)));
    checkCudaErrors(hipMemset(gridFlowHash[dev],-1,lenCell*sizeof(int)));
  }

gaussian_array_initH();
domInfo_array_initH();
}

texture<float,1,hipReadModeElementType> texRefGaussian;
texture<int,1,hipReadModeElementType> texRefDomInfo;

void domInfo_array_initH()
{
//ref the structure of grid_info in domain.h
int indexTypeLen=3;//0~2 correspond to is,ie,in  if indexDir<3; 0~2 correspond to s1,s2,s3 if indexDir=3
int incGhostLen=2;//0~1 correspond to include ghost cell or not
int indexDirLen=4;//0~2 correspond to i,j,k
int hostDeviceLen=2; //0~1 correspond to host,device
int coordiSysLen=4; //0~3 correspond to Gcc,Gfx,Gfy,Gfz.
int domInfo_array_len=coordiSysLen*hostDeviceLen*indexDirLen*incGhostLen*indexTypeLen;
 
//allocate memory for DomInfo and _DomInfo
DomInfo=(int *) malloc(sizeof(int) * domInfo_array_len);
checkCudaErrors(hipMalloc((void**) &(_DomInfo),sizeof(int) * domInfo_array_len));

grid_info G;
int indexType=3;//0~2 correspond to is,ie,in  if indexDir<3; 0~2 correspond to s1,s2,s3 if indexDir=3
int incGhost=2;//0~1 correspond to include ghost cell or not
int indexDir=4;//0~2 correspond to i,j,k
int hostDevice=2; //0~1 correspond to host,device
int coordiSys=4; //0~3 correspond to Gcc,Gfx,Gfy,Gfz.
int index; 
//coordiSys= 0~3 correspond to Gcc,Gfx,Gfy,Gfz.
for(coordiSys=0;coordiSys<coordiSysLen;coordiSys++)
{
switch(coordiSys)
{
case 0:G=Dom.Gcc;break;
case 1:G=Dom.Gfx;break;
case 2:G=Dom.Gfy;break;
case 3:G=Dom.Gfz;break;
default: break;
}

hostDevice=0;
indexDir=0;
incGhost=0;
indexType=0;
index=indexType+incGhost*3+indexDir*6+ hostDevice*24+ coordiSys*48;

DomInfo[index]  =G.is;
DomInfo[index+1]=G.ie;
DomInfo[index+2]=G.in;
DomInfo[index+3]=G.isb;
DomInfo[index+4]=G.ieb;
DomInfo[index+5]=G.inb;

DomInfo[index+6] =G.js;
DomInfo[index+7] =G.je;
DomInfo[index+8] =G.jn;
DomInfo[index+9] =G.jsb;
DomInfo[index+10]=G.jeb;
DomInfo[index+11]=G.jnb;

DomInfo[index+12] =G.ks;
DomInfo[index+13] =G.ke;
DomInfo[index+14] =G.kn;
DomInfo[index+15] =G.ksb;
DomInfo[index+16] =G.keb;
DomInfo[index+17] =G.knb;

DomInfo[index+18] =G.s1;
DomInfo[index+19] =G.s2;
DomInfo[index+20] =G.s3;
DomInfo[index+21] =G.s1b;
DomInfo[index+22] =G.s2b;
DomInfo[index+23] =G.s3b;
//printf("\nDomInfo: %d %d %d %d\n",G.s1b,G.s2b,G.s3b,coordiSys);
DomInfo[index+24]=G._is;
DomInfo[index+25]=G._ie;
DomInfo[index+26]=G._in;
DomInfo[index+27]=G._isb;
DomInfo[index+28]=G._ieb;
DomInfo[index+29]=G._inb;

DomInfo[index+30]=G._js;
DomInfo[index+31]=G._je;
DomInfo[index+32]=G._jn;
DomInfo[index+33]=G._jsb;
DomInfo[index+34]=G._jeb;
DomInfo[index+35]=G._jnb;

DomInfo[index+36] =G._ks;
DomInfo[index+37] =G._ke;
DomInfo[index+38] =G._kn;
DomInfo[index+39] =G._ksb;
DomInfo[index+40] =G._keb;
DomInfo[index+41] =G._knb;

DomInfo[index+42] =G._s1;
DomInfo[index+43] =G._s2;
DomInfo[index+44] =G._s3;
DomInfo[index+45] =G._s1b;
DomInfo[index+46] =G._s2b;
DomInfo[index+47] =G._s3b;
//printf("\ndevice DomInfo: %d %d %d %d\n",G._s1b,G._s2b,G._s3b,coordiSys); //_s1b~_s3b =0 at this time
}


checkCudaErrors(hipMemcpy(_DomInfo, DomInfo, sizeof(int) * domInfo_array_len,
      hipMemcpyHostToDevice));
hipBindTexture(0,texRefDomInfo,_DomInfo,sizeof(int)*domInfo_array_len);

}

void gaussian_array_initH()
{
//put into malloc and free
//    int LEN_GAUSSIAN_ARRAY=5000;
    //texture<float,1,hipReadModeElementType> texRefGaussian;

    checkCudaErrors(hipMalloc((void**) &(GaussianKernel),sizeof(float) * LEN_GAUSSIAN_ARRAY));

    dim3 dimBlocks_p,numBlocks_p;
    block_thread_point(dimBlocks_p,numBlocks_p,LEN_GAUSSIAN_ARRAY);


real dx=Dom.dx;real dy=Dom.dy;real dz=Dom.dz;    
real min_meshsize=min(min(dx,dy),dz);
real max_meshsize=max(max(dx,dy),dz);




//real min_meshsize=min(min(dx,dy),dz);
//2.0f*sqrt(2.0f*log(2.0f))=2.3548f;
real sig= min_meshsize/2.3548f;
real norm=pow(sqrt(2*PI)*sig,3);
real maxLenR=2.6f*max_meshsize;
real maxLenR2=maxLenR*maxLenR;
//real dg=maxLenR/(float)(LEN_GAUSSIAN_ARRAY);

real dg2=maxLenR2/(float)(LEN_GAUSSIAN_ARRAY);

//real dg2_sig2=dg*dg/2.f/sig/sig;
real dg2_sig2=dg2/2.f/sig/sig;

    gaussian_array_initD<<<numBlocks_p,dimBlocks_p>>>(GaussianKernel,dg2_sig2,dg2,norm);
fflush(stdout);
    hipBindTexture(0,texRefGaussian,GaussianKernel,sizeof(float)*LEN_GAUSSIAN_ARRAY);
}


void cuda_vorticity(int dev)
{

dim3 dimBlocks,numBlocks;
dim3 dimBlocks_u,numBlocks_u;
dim3 dimBlocks_v,numBlocks_v;
dim3 dimBlocks_w,numBlocks_w;

int coordiSys,planeDirc;

coordiSys=0;
planeDirc=1;
block_thread_cell_noOverLap(dimBlocks,numBlocks,dom[dev],coordiSys,planeDirc);

coordiSys=1;
planeDirc=coordiSys;
block_thread_cell(dimBlocks_u,numBlocks_u,dom[dev],coordiSys,planeDirc);
coordiSys=2;
planeDirc=coordiSys;
block_thread_cell(dimBlocks_v,numBlocks_v,dom[dev],coordiSys,planeDirc);
coordiSys=3;
planeDirc=coordiSys;
block_thread_cell(dimBlocks_w,numBlocks_w,dom[dev],coordiSys,planeDirc);

real *dudy,*dudz;
real *dvdx,*dvdz;
real *dwdx,*dwdy;

 checkCudaErrors(hipMalloc((void**) &dudy, 
	sizeof(real) * dom[dev].Gcc.s3b));
 checkCudaErrors(hipMalloc((void**) &dudz, 
	sizeof(real) * dom[dev].Gcc.s3b));
 checkCudaErrors(hipMalloc((void**) &dvdx, 
	sizeof(real) * dom[dev].Gcc.s3b));
 checkCudaErrors(hipMalloc((void**) &dvdz, 
	sizeof(real) * dom[dev].Gcc.s3b));
 checkCudaErrors(hipMalloc((void**) &dwdx, 
	sizeof(real) * dom[dev].Gcc.s3b));
 checkCudaErrors(hipMalloc((void**) &dwdy, 
	sizeof(real) * dom[dev].Gcc.s3b));


gradU<<<numBlocks_u,dimBlocks_u>>>(_u[dev],dudy,dudz,_dom[dev]);
gradV<<<numBlocks_v,dimBlocks_v>>>(_v[dev],dvdx,dvdz,_dom[dev]);
gradW<<<numBlocks_w,dimBlocks_w>>>(_w[dev],dwdx,dwdy,_dom[dev]);


Omega<<<numBlocks,dimBlocks>>>(_omega_x[dev],_omega_y[dev],_omega_z[dev], 
			 dudy, dudz,
			 dvdx, dvdz,
			 dwdx, dwdy,
			 _dom[dev]);

hipFree(dudy);
hipFree(dudz);

hipFree(dvdx);
hipFree(dvdz);

hipFree(dwdx);
hipFree(dwdy);
}







//extern "C"
void lpt_point_source_mollify_final()
{
hipFree(GaussianKernel);
hipUnbindTexture(texRefGaussian);

free(DomInfo);
hipFree(_DomInfo);
hipUnbindTexture(texRefDomInfo);

cuda_free_array_real(posX);
cuda_free_array_real(posY);
cuda_free_array_real(posZ);

cuda_free_array_real(posXold);
cuda_free_array_real(posYold);
cuda_free_array_real(posZold);

cuda_free_array_real(lptSourceVal);
cuda_free_array_real(lptSourceValOld);

cuda_free_array_real(ug);
cuda_free_array_real(vg);
cuda_free_array_real(wg);

cuda_free_array_real(scg);
cuda_free_array_real(Ksi);
//cuda_free_array_real(Weight);

cuda_free_array_real(lpt_stress_u);
cuda_free_array_real(lpt_stress_v);
cuda_free_array_real(lpt_stress_w);

cuda_free_array_real(lpt_omegaX);
cuda_free_array_real(lpt_omegaY);
cuda_free_array_real(lpt_omegaZ);


if(lpt_twoway>0)
{
cuda_free_array_real(lpt_dudt);
cuda_free_array_real(lpt_dvdt);
cuda_free_array_real(lpt_dwdt);
}

cuda_free_array_int(cellStart);
cuda_free_array_int(cellEnd);
cuda_free_array_int(pointNumInCell);
cuda_free_array_int(gridFlowHash);

cuda_free_array_int(gridParticleIndex);
cuda_free_array_int(gridParticleHash);
}



extern "C"
void cuda_build_cages(void)
{
  cuda_point_pull();

  // parallelize over domains
  #pragma omp parallel num_threads(nsubdom)
  {
    int dev = omp_get_thread_num();
    checkCudaErrors(hipSetDevice(dev + dev_start));

    int threads_x = 0;
    int threads_y = 0;
    int threads_z = 0;
    int blocks_x = 0;
    int blocks_y = 0;
    int blocks_z = 0;


    // reset flag_u
    if(dom[dev].Gfx.jn < MAX_THREADS_DIM)
      threads_y = dom[dev].Gfx.jnb;
    else
      threads_y = MAX_THREADS_DIM;

    if(dom[dev].Gfx.kn < MAX_THREADS_DIM)
      threads_z = dom[dev].Gfx.knb;
    else
      threads_z = MAX_THREADS_DIM;

    blocks_y = (int)ceil((real) dom[dev].Gfx.jnb / (real) threads_y);
    blocks_z = (int)ceil((real) dom[dev].Gfx.knb / (real) threads_z);

    dim3 dimBlocks_u(threads_y, threads_z);
    dim3 numBlocks_u(blocks_y, blocks_z);

    reset_flag_u<<<numBlocks_u, dimBlocks_u>>>(_flag_u[dev], _dom[dev], bc);

    // reset flag_v
    if(dom[dev].Gfy.kn < MAX_THREADS_DIM)
      threads_z = dom[dev].Gfy.knb;
    else
      threads_z = MAX_THREADS_DIM;

    if(dom[dev].Gfy.in < MAX_THREADS_DIM)
      threads_x = dom[dev].Gfy.inb;
    else
      threads_x = MAX_THREADS_DIM;

    blocks_z = (int)ceil((real) dom[dev].Gfy.knb / (real) threads_z);
    blocks_x = (int)ceil((real) dom[dev].Gfy.inb / (real) threads_x);

    dim3 dimBlocks_v(threads_z, threads_x);
    dim3 numBlocks_v(blocks_z, blocks_x);

    reset_flag_v<<<numBlocks_v, dimBlocks_v>>>(_flag_v[dev], _dom[dev], bc);

    // reset flag_w
    if(dom[dev].Gfz.in < MAX_THREADS_DIM)
      threads_x = dom[dev].Gfz.inb;
    else
      threads_x = MAX_THREADS_DIM;

    if(dom[dev].Gfz.jn < MAX_THREADS_DIM)
      threads_y = dom[dev].Gfz.jnb;
    else
      threads_y = MAX_THREADS_DIM;

    blocks_x = (int)ceil((real) dom[dev].Gfz.inb / (real) threads_x);
    blocks_y = (int)ceil((real) dom[dev].Gfz.jnb / (real) threads_y);

    dim3 dimBlocks_w(threads_x, threads_y);
    dim3 numBlocks_w(blocks_x, blocks_y);

    reset_flag_w<<<numBlocks_w, dimBlocks_w>>>(_flag_w[dev], _dom[dev], bc);


 // push point_particle information to device
    checkCudaErrors(hipMemcpy(_points[dev], points, sizeof(point_struct) * npoints,
      hipMemcpyHostToDevice));



  
  
      // fill in ghost cells for periodic boundary conditions
      if(bc.uW == PERIODIC) {
        blocks_y = (int)ceil((real) dom[dev].Gfx.jnb / (real) threads_y);
        blocks_z = (int)ceil((real) dom[dev].Gfx.knb / (real) threads_z);
        numBlocks_u.x = blocks_y;
        numBlocks_u.y = blocks_z;
        cage_flag_u_periodic_W<<<numBlocks_u, dimBlocks_u>>>(_flag_u[dev],
          _dom[dev]);
      }
      if(bc.uE == PERIODIC) {
        blocks_y = (int)ceil((real) dom[dev].Gfx.jnb / (real) threads_y);
        blocks_z = (int)ceil((real) dom[dev].Gfx.knb / (real) threads_z);
        numBlocks_u.x = blocks_y;
        numBlocks_u.y = blocks_z;
        cage_flag_u_periodic_E<<<numBlocks_u, dimBlocks_u>>>(_flag_u[dev],
          _dom[dev]);
      }
      if(bc.uS == PERIODIC) {
        blocks_z = (int)ceil((real) dom[dev].Gfx.knb / (real) threads_z);
        blocks_x = (int)ceil((real) dom[dev].Gfx.inb / (real) threads_x);
        numBlocks_u.x = blocks_z;
        numBlocks_u.y = blocks_x;
        cage_flag_u_periodic_S<<<numBlocks_u, dimBlocks_u>>>(_flag_u[dev],
          _dom[dev]);
      }
      if(bc.uN == PERIODIC) {
        blocks_z = (int)ceil((real) dom[dev].Gfx.knb / (real) threads_z);
        blocks_x = (int)ceil((real) dom[dev].Gfx.inb / (real) threads_x);
        numBlocks_u.x = blocks_z;
        numBlocks_u.y = blocks_x;
        cage_flag_u_periodic_N<<<numBlocks_u, dimBlocks_u>>>(_flag_u[dev],
          _dom[dev]);
      }
      if(bc.uB == PERIODIC) {
        blocks_x = (int)ceil((real) dom[dev].Gfx.inb / (real) threads_x);
        blocks_y = (int)ceil((real) dom[dev].Gfx.jnb / (real) threads_y);
        numBlocks_u.x = blocks_x;
        numBlocks_u.y = blocks_y;
        cage_flag_u_periodic_B<<<numBlocks_u, dimBlocks_u>>>(_flag_u[dev],
          _dom[dev]);
      }
      if(bc.uT == PERIODIC) {
        blocks_x = (int)ceil((real) dom[dev].Gfx.inb / (real) threads_x);
        blocks_y = (int)ceil((real) dom[dev].Gfx.jnb / (real) threads_y);
        numBlocks_u.x = blocks_x;
        numBlocks_u.y = blocks_y;
        cage_flag_u_periodic_T<<<numBlocks_u, dimBlocks_u>>>(_flag_u[dev],
          _dom[dev]);
      }



      if(bc.vW == PERIODIC) {
        blocks_y = (int)ceil((real) dom[dev].Gfy.jnb / (real) threads_y);
        blocks_z = (int)ceil((real) dom[dev].Gfy.knb / (real) threads_z);
        numBlocks_v.x = blocks_y;
        numBlocks_v.y = blocks_z;
        cage_flag_v_periodic_W<<<numBlocks_v, dimBlocks_v>>>(_flag_v[dev],
          _dom[dev]);
      }
      if(bc.vE == PERIODIC) {
        blocks_y = (int)ceil((real) dom[dev].Gfy.jnb / (real) threads_y);
        blocks_z = (int)ceil((real) dom[dev].Gfy.knb / (real) threads_z);
        numBlocks_v.x = blocks_y;
        numBlocks_v.y = blocks_z;
        cage_flag_v_periodic_E<<<numBlocks_v, dimBlocks_v>>>(_flag_v[dev],
          _dom[dev]);
      }
      if(bc.vS == PERIODIC) {
        blocks_z = (int)ceil((real) dom[dev].Gfy.knb / (real) threads_z);
        blocks_x = (int)ceil((real) dom[dev].Gfy.inb / (real) threads_x);
        numBlocks_v.x = blocks_z;
        numBlocks_v.y = blocks_x;
        cage_flag_v_periodic_S<<<numBlocks_v, dimBlocks_v>>>(_flag_v[dev],
          _dom[dev]);
      }
      if(bc.vN == PERIODIC) {
        blocks_z = (int)ceil((real) dom[dev].Gfy.knb / (real) threads_z);
        blocks_x = (int)ceil((real) dom[dev].Gfy.inb / (real) threads_x);
        numBlocks_v.x = blocks_z;
        numBlocks_v.y = blocks_x;
        cage_flag_v_periodic_N<<<numBlocks_v, dimBlocks_v>>>(_flag_v[dev],
          _dom[dev]);
      }
      if(bc.vB == PERIODIC) {
        blocks_x = (int)ceil((real) dom[dev].Gfy.inb / (real) threads_x);
        blocks_y = (int)ceil((real) dom[dev].Gfy.jnb / (real) threads_y);
        numBlocks_v.x = blocks_x;
        numBlocks_v.y = blocks_y;
        cage_flag_v_periodic_B<<<numBlocks_v, dimBlocks_v>>>(_flag_v[dev],
          _dom[dev]);
      }
      if(bc.vT == PERIODIC) {
        blocks_x = (int)ceil((real) dom[dev].Gfy.inb / (real) threads_x);
        blocks_y = (int)ceil((real) dom[dev].Gfy.jnb / (real) threads_y);
        numBlocks_v.x = blocks_x;
        numBlocks_v.y = blocks_y;
        cage_flag_v_periodic_T<<<numBlocks_v, dimBlocks_v>>>(_flag_v[dev],
          _dom[dev]);
      }




      if(bc.wW == PERIODIC) {
        blocks_y = (int)ceil((real) dom[dev].Gfz.jnb / (real) threads_y);
        blocks_z = (int)ceil((real) dom[dev].Gfz.knb / (real) threads_z);
        numBlocks_w.x = blocks_y;
        numBlocks_w.y = blocks_z;
        cage_flag_w_periodic_W<<<numBlocks_w, dimBlocks_w>>>(_flag_w[dev],
          _dom[dev]);
      }
      if(bc.wE == PERIODIC) {
        blocks_y = (int)ceil((real) dom[dev].Gfz.jnb / (real) threads_y);
        blocks_z = (int)ceil((real) dom[dev].Gfz.knb / (real) threads_z);
        numBlocks_w.x = blocks_y;
        numBlocks_w.y = blocks_z;
        cage_flag_w_periodic_E<<<numBlocks_w, dimBlocks_w>>>(_flag_w[dev],
          _dom[dev]);
      }
      if(bc.wS == PERIODIC) {
        blocks_z = (int)ceil((real) dom[dev].Gfz.knb / (real) threads_z);
        blocks_x = (int)ceil((real) dom[dev].Gfz.inb / (real) threads_x);
        numBlocks_w.x = blocks_z;
        numBlocks_w.y = blocks_x;
        cage_flag_w_periodic_S<<<numBlocks_w, dimBlocks_w>>>(_flag_w[dev],
          _dom[dev]);
      }
      if(bc.wN == PERIODIC) {
        blocks_z = (int)ceil((real) dom[dev].Gfz.knb / (real) threads_z);
        blocks_x = (int)ceil((real) dom[dev].Gfz.inb / (real) threads_x);
        numBlocks_w.x = blocks_z;
        numBlocks_w.y = blocks_x;
        cage_flag_w_periodic_N<<<numBlocks_w, dimBlocks_w>>>(_flag_w[dev],
          _dom[dev]);
      }
      if(bc.wB == PERIODIC) {
        blocks_x = (int)ceil((real) dom[dev].Gfz.inb / (real) threads_x);
        blocks_y = (int)ceil((real) dom[dev].Gfz.jnb / (real) threads_y);
        numBlocks_w.x = blocks_x;
        numBlocks_w.y = blocks_y;
        cage_flag_w_periodic_B<<<numBlocks_w, dimBlocks_w>>>(_flag_w[dev],
          _dom[dev]);
      }
      if(bc.wT == PERIODIC) {
        blocks_x = (int)ceil((real) dom[dev].Gfz.inb / (real) threads_x);
        blocks_y = (int)ceil((real) dom[dev].Gfz.jnb / (real) threads_y);
        numBlocks_w.x = blocks_x;
        numBlocks_w.y = blocks_y;
        cage_flag_w_periodic_T<<<numBlocks_w, dimBlocks_w>>>(_flag_w[dev],
          _dom[dev]);
                           }
   }
  
}


