#include "hip/hip_runtime.h"
/*
extern "C"
{
#include "bluebottle.h"
#include "stdio.h"
#include "hip/hip_runtime.h"
//#include "cuda_scalar.h"
}
*/
#include "cuda_scalar.h"
#include "cuda_point.h"

__global__ void array_init(real *A,dom_struct *dom, int n, real a)
{
int indx =  threadIdx.x + blockIdx.x*blockDim.x;
int indy =  threadIdx.y + blockIdx.y*blockDim.y;
int pp=indx+indy*gridDim.x*blockDim.x;
if(pp<n) A[pp]=a;
}



/*
A is fluid property located at cell center or face center depending on dirc 
dirc=0 coresponds to cell-center; dirc=1 x-face-center;dirc=2 y-face-center;dirc=3 z-face-center
dir
*/
__global__ void lpt_mollify_sc(int npoints, real *A, point_struct *points, dom_struct *dom,real *Ksi, int dirc, int dirc2)
{

int indx =  threadIdx.x + blockIdx.x*blockDim.x;
int indy =  threadIdx.y + blockIdx.y*blockDim.y;
int pp=indx+indy*gridDim.x*blockDim.x;

//if(pp<npoints*STENCIL3)  Ksi[pp]=0.0;
//__sycthreads();

if(pp<npoints)
{

////printf("\npp indx %d %d %d \n",pp, indx, indy);
////printf("\nblockDim.x,gridDim.x %d %d %d %d\n",blockDim.x,gridDim.x,blockDim.y,gridDim.y );

//Define the STENCIL of the Gausian filter, the spread length of Gaussian kernel
real ksi[STENCIL][STENCIL][STENCIL];
real xs=dom->xs;real ys=dom->ys;real zs=dom->zs;
real dx=dom->dx;real dy=dom->dy;real dz=dom->dz;
  real ddx = 1. / dom->dx;
  real ddy = 1. / dom->dy;
  real ddz = 1. / dom->dz;
real cellVol=dx*dy*dz;

real  xp =  points[pp].x;
real  yp =  points[pp].y;
real  zp =  points[pp].z;

// i,j,k should be the start number of cell center, so that xm(i)<=xp<xm(i+1)
//xm(i)= (i-0.5)*dom->dx +dom->xs
//i=floor((xm - dom->xs) * ddx+ 0.5);
//i=floor((xm - dom->xs) * ddx- 0.5) + DOM_BUF;
  int ip,jp,kp;
  int is,js,ks;

//The cell-center or face-center index to locate the particle 
switch(dirc)
{
case 0:
  ip = floor((xp - dom->xs) * ddx) + DOM_BUF;//for xm[i]
  jp = floor((yp - dom->ys) * ddy) + DOM_BUF;//for ym[j]
  kp = floor((zp - dom->zs) * ddz) + DOM_BUF;//for zm[k]
  break;
case 1:
  ip = floor((xp - dom->xs) * ddx + 0.5) + DOM_BUF;//for xm[i+1], ip should be i+1
  jp = floor((yp - dom->ys) * ddy) + DOM_BUF;	   //for y[i]
  kp = floor((zp - dom->zs) * ddz) + DOM_BUF;	   //for z[i]
  break;
case 2:
  ip = floor((xp - dom->xs) * ddx) + DOM_BUF;//for x[i] 
  jp = floor((yp - dom->ys) * ddy + 0.5) + DOM_BUF;	   //for ym[i+1]
  kp = floor((zp - dom->zs) * ddz) + DOM_BUF;	   //for z[i]
  break;
case 3:
  ip = floor((xp - dom->xs) * ddx) + DOM_BUF;//for x[i] 
  jp = floor((yp - dom->ys) * ddy) + DOM_BUF;	   //for y[i]
  kp = floor((zp - dom->zs) * ddz+ 0.5) + DOM_BUF;  //for zm[i+1]
  break;
default:
  printf("Wrong direction in lpt_mollify");
}

points[pp].i=ip;
points[pp].j=jp;
points[pp].k=kp;

real buf=0;
int began=- round((STENCIL-1)/2.0f);//equal to -1 if STENCIL=3
int end=1+ ceil((STENCIL-1)/2.0f);//equal to 2 if STENCIL=3


//Calculate the filter strength from Gaussian kernel
for(int dk=began;dk<end;dk++)
for(int dj=began;dj<end;dj++)
for(int di=began;di<end;di++)
{
is=di-began;js=dj-began;ks=dk-began;
ksi[is][js][ks]=lpt_integrate_mol(ip+di,jp+dj,kp+dk,xp,yp,zp,dx,dy,dz,xs,ys,zs,dirc);
buf+=ksi[is][js][ks];
}

//TODO add mask infomation as in lpt_mollify_sc in lpt_interpolator.f90

real Ap;
switch(dirc)
{
case 0:
    switch(dirc2)
   { case 0:   
//Exchange rate of soluble mass from particles. All the following source should divide cellVol to ensure conservation law
     Ap=-points[pp].msdot/cellVol;break;
//     Ap=1/cellVol;break;

     case 1:
//Particle volume filter
     Ap=PI*4/3*points[pp].r*points[pp].r*points[pp].r/cellVol;break;
     default: Ap=0;break;
	}
break;
case 1:
//Particle x-momentum reaction to fluid
Ap=-points[pp].Fx/cellVol;
break;
case 2:
Ap=-points[pp].Fy/cellVol;
break;
case 3:
//Particle z-momentum reaction to fluid
Ap=-points[pp].Fz/cellVol;
break;
default:
break;
	}		


// Normalize  ksi = ksi/buf
if(buf>0)
	{
for(int dk=began;dk<end;dk++)
for(int dj=began;dj<end;dj++)
for(int di=began;di<end;di++)
							{
is=di-began;js=dj-began;ks=dk-began;
ksi[is][js][ks]=ksi[is][js][ks]/buf;
Ksi[is+js*STENCIL+ks*STENCIL2+pp*STENCIL3]=ksi[is][js][ks]*Ap; }
		}


	}

}


//Add the point source to fluid source weighted by the Gaussian filter coefficient Ksi
__global__ void lpt_sum_ksi(int npoints, real *A, point_struct *points, dom_struct *dom,real *Ksi, int dirc,int dirc2)
{
int indx =  threadIdx.x + blockIdx.x*blockDim.x;
int indy =  threadIdx.y + blockIdx.y*blockDim.y;
int np=indx+indy*gridDim.x*blockDim.x;
if(np<1)
{

int began=-round((STENCIL-1)/2.0);//equal to -1 if STENCIL=3
int end=1+ ceil((STENCIL-1)/2.0);//equal to 2 if STENCIL=3

//printf("\nST %d %d\n",end-began+1,STENCIL);
  int ip,jp,kp;
  int is,js,ks;
  int ic,jc,kc;
real ksi;

for(int pp=0;pp<npoints;pp++)
{
ip =  points[pp].i;
jp =  points[pp].j;
kp =  points[pp].k;

// Perform the actual extrapolation on A
for(int dk=began;dk<end;dk++)
for(int dj=began;dj<end;dj++)
for(int di=began;di<end;di++)
{
  ic=di+ip;jc=dj+jp;kc=dk+kp;
  is=di-began;js=dj-began;ks=dk-began;
  ksi=Ksi[is+js*STENCIL+ks*STENCIL2+pp*STENCIL3];
switch(dirc)
{
case 0:
//Make the index inside the domain before we change the property at this location
periodic_grid_index(ic,jc,kc,dom,0);
A[ic+jc*dom->Gcc._s1b+kc*dom->Gcc._s2b]+=ksi;
break;
case 1:
periodic_grid_index(ic,jc,kc,dom,1);
A[ic+jc*dom->Gfx._s1b+kc*dom->Gfx._s2b]+=ksi;
break;
case 2:
periodic_grid_index(ic,jc,kc,dom,2);
A[ic+jc*dom->Gfy._s1b+kc*dom->Gfy._s2b]+=ksi;
break;
case 3:
periodic_grid_index(ic,jc,kc,dom,3);
A[ic+jc*dom->Gfz._s1b+kc*dom->Gfz._s2b]+=ksi;
break;
default:
break;
			}		

		}
	}
  }
}



/*
__global__ void lpt_mollify_sc(int npoints, real *A, point_struct *points, dom_struct *dom,real *Ksi, int dirc, int dirc2)
{

int indx =  threadIdx.x + blockIdx.x*blockDim.x;
int indy =  threadIdx.y + blockIdx.y*blockDim.y;
int pp=indx+indy*gridDim.x*blockDim.x;

//if(pp<npoints*STENCIL3)  Ksi[pp]=0.0;
//__sycthreads();

if(pp<npoints)
{

////printf("\npp indx %d %d %d \n",pp, indx, indy);
////printf("\nblockDim.x,gridDim.x %d %d %d %d\n",blockDim.x,gridDim.x,blockDim.y,gridDim.y );

//Define the STENCIL of the Gausian filter, the spread length of Gaussian kernel
real ksi[STENCIL][STENCIL][STENCIL];
real xs=dom->xs;real ys=dom->ys;real zs=dom->zs;
real dx=dom->dx;real dy=dom->dy;real dz=dom->dz;
  real ddx = 1. / dom->dx;
  real ddy = 1. / dom->dy;
  real ddz = 1. / dom->dz;
//real cellVol=dx*dy*dz;

real  xp =  points[pp].x;
real  yp =  points[pp].y;
real  zp =  points[pp].z;

// i,j,k should be the start number of cell center, so that xm(i)<=xp<xm(i+1)
//xm(i)= (i-0.5)*dom->dx +dom->xs
//i=floor((xm - dom->xs) * ddx+ 0.5);
//i=floor((xm - dom->xs) * ddx- 0.5) + DOM_BUF;
  int ip,jp,kp;
  int is,js,ks;

//The cell-center or face-center index to locate the particle 
switch(dirc)
{
case 0:
  ip = floor((xp - dom->xs) * ddx) + DOM_BUF;//for xm[i]
  jp = floor((yp - dom->ys) * ddy) + DOM_BUF;//for ym[j]
  kp = floor((zp - dom->zs) * ddz) + DOM_BUF;//for zm[k]
  break;
case 1:
  ip = floor((xp - dom->xs) * ddx + 0.5) + DOM_BUF;//for xm[i+1], ip should be i+1
  jp = floor((yp - dom->ys) * ddy) + DOM_BUF;	   //for y[i]
  kp = floor((zp - dom->zs) * ddz) + DOM_BUF;	   //for z[i]
  break;
case 2:
  ip = floor((xp - dom->xs) * ddx) + DOM_BUF;//for x[i] 
  jp = floor((yp - dom->ys) * ddy + 0.5) + DOM_BUF;	   //for ym[i+1]
  kp = floor((zp - dom->zs) * ddz) + DOM_BUF;	   //for z[i]
  break;
case 3:
  ip = floor((xp - dom->xs) * ddx) + DOM_BUF;//for x[i] 
  jp = floor((yp - dom->ys) * ddy) + DOM_BUF;	   //for y[i]
  kp = floor((zp - dom->zs) * ddz+ 0.5) + DOM_BUF;  //for zm[i+1]
  break;
default:
  printf("Wrong direction in lpt_mollify");
}

points[pp].i=ip;
points[pp].j=jp;
points[pp].k=kp;

real buf=0;
int began=-(STENCIL-1)/2;//equal to -1 if STENCIL=3
int end=1+(STENCIL-1)/2;//equal to 2 if STENCIL=3


//Calculate the filter strength from Gaussian kernel
for(int dk=began;dk<end;dk++)
for(int dj=began;dj<end;dj++)
for(int di=began;di<end;di++)
{
is=di-began;js=dj-began;ks=dk-began;
ksi[is][js][ks]=lpt_integrate_mol(ip+di,jp+dj,kp+dk,xp,yp,zp,dx,dy,dz,xs,ys,zs,dirc);
buf+=ksi[is][js][ks];
}

//TODO add mask infomation as in lpt_mollify_sc in lpt_interpolator.f90

// Normalize  ksi = ksi/buf
if(buf>0)
{
for(int dk=began;dk<end;dk++)
for(int dj=began;dj<end;dj++)
for(int di=began;di<end;di++)
	{
is=di-began;js=dj-began;ks=dk-began;
ksi[is][js][ks]=ksi[is][js][ks]/buf;
Ksi[is+js*STENCIL+ks*STENCIL2+pp*STENCIL3]=ksi[is][js][ks];
			}
		}


	}

}


//Add the point source to fluid source weighted by the Gaussian filter coefficient Ksi
__global__ void lpt_sum_ksi(int npoints, real *A, point_struct *points, dom_struct *dom,real *Ksi, int dirc,int dirc2)
{
int indx =  threadIdx.x + blockIdx.x*blockDim.x;
int indy =  threadIdx.y + blockIdx.y*blockDim.y;
int np=indx+indy;
if(np<1)
{

int began=-(STENCIL-1)/2;//equal to -1 if STENCIL=3
int end=1+(STENCIL-1)/2;//equal to 2 if STENCIL=3
  int ip,jp,kp;
  int is,js,ks;
  int ic,jc,kc;
real Ap,ksi;
real cellVol=dom->dx*dom->dy*dom->dz;

for(int pp=0;pp<npoints;pp++)
{

ip =  points[pp].i;
jp =  points[pp].j;
kp =  points[pp].k;

// Perform the actual extrapolation on A
for(int dk=began;dk<end;dk++)
for(int dj=began;dj<end;dj++)
for(int di=began;di<end;di++)
{
  ic=di+ip;jc=dj+jp;kc=dk+kp;
  is=di-began;js=dj-began;ks=dk-began;
  ksi=Ksi[is+js*STENCIL+ks*STENCIL2+pp*STENCIL3];
switch(dirc)
{
case 0:
//Make the index inside the domain before we change the property at this location
periodic_grid_index(ic,jc,kc,dom,0);
    switch(dirc2)
   { case 0:   
//Exchange rate of soluble mass from particles. All the following source should divide cellVol to ensure conservation law
     Ap=-points[pp].msdot/cellVol;break;
//    Ap=1/cellVol;break;
     case 1:
//Particle volume filter
     Ap=PI*4/3*points[pp].r*points[pp].r*points[pp].r/cellVol;break;
     default:
printf("Wrong direction in cell center mollify");
	}
 A[ic+jc*dom->Gcc._s1b+kc*dom->Gcc._s2b]+=ksi*Ap;
break;
case 1:
periodic_grid_index(ic,jc,kc,dom,1);

//Particle x-momentum reaction to fluid
Ap=-points[pp].Fx/cellVol;
//Ap=1/cellVol;
//Ap=0;
A[ic+jc*dom->Gfx._s1b+kc*dom->Gfx._s2b]+=ksi*Ap;
break;
case 2:
periodic_grid_index(ic,jc,kc,dom,2);
//Particle y-momentum reaction to fluid
Ap=-points[pp].Fy/cellVol;
//Ap=1/cellVol;
//Ap=0;
A[ic+jc*dom->Gfy._s1b+kc*dom->Gfy._s2b]+=ksi*Ap;
break;
case 3:
periodic_grid_index(ic,jc,kc,dom,3);
//Particle z-momentum reaction to fluid
Ap=-points[pp].Fz/cellVol;
//Ap=1/cellVol;
//Ap=0;
A[ic+jc*dom->Gfz._s1b+kc*dom->Gfz._s2b]+=ksi*Ap;

break;
default:
break;
			}		

		}
	}
  }
}

*/


__device__ void periodic_grid_index(int ic,int jc,int kc,dom_struct *dom, int dirc)
{
//int tag=0;
switch(dirc)
{
case 0:
if(ic<dom->Gcc.is||ic > dom->Gcc.ie-1)  ic=(int) fmodf(ic,dom->Gcc.in);
if(jc<dom->Gcc.js||jc > dom->Gcc.je-1)  jc=(int) fmodf(jc,dom->Gcc.jn);
if(kc<dom->Gcc.ks||kc > dom->Gcc.ke-1)  kc=(int) fmodf(kc,dom->Gcc.kn);
//after fmod, the value could still be negative
  if(ic < dom->Gcc.is) ic = ic +dom->Gcc.in;
  if(jc < dom->Gcc.js) jc = jc +dom->Gcc.jn;
  if(kc < dom->Gcc.ks) kc = kc +dom->Gcc.kn;
 break;
case 1:
if(ic<dom->Gfx.is||ic > dom->Gfx.ie-2)  ic=(int) fmodf(ic,dom->Gfx.in-1);
if(jc<dom->Gfx.js||jc > dom->Gfx.je-1)  jc=(int) fmodf(jc,dom->Gfx.jn);
if(kc<dom->Gfx.ks||kc > dom->Gfx.ke-1)  kc=(int) fmodf(kc,dom->Gfx.kn);
  if(ic < dom->Gfx.is) ic = ic+(dom->Gfx.in-1);
  if(jc < dom->Gfx.js) jc = jc+dom->Gfx.jn;
  if(kc < dom->Gfx.ks) kc = kc+dom->Gfx.kn;
break;
case 2:
if(ic<dom->Gfy.is||ic > dom->Gfy.ie-1)  ic=(int) fmodf(ic,dom->Gfy.in);
if(jc<dom->Gfy.js||jc > dom->Gfy.je-2)  jc=(int) fmodf(jc,dom->Gfy.jn-1);
if(kc<dom->Gfy.ks||kc > dom->Gfy.ke-1)  kc=(int) fmodf(kc,dom->Gfy.kn);
  if(ic < dom->Gfy.is) ic = ic+dom->Gfy.in;
  if(jc < dom->Gfy.js) jc = jc+(dom->Gfy.jn-1);
  if(kc < dom->Gfy.ks) kc = kc+dom->Gfy.kn;
break;
case 3:
if(ic<dom->Gfz.is||ic > dom->Gfz.ie-1)  ic=(int) fmodf(ic,dom->Gfz.in);
if(jc<dom->Gfz.js||jc > dom->Gfz.je-1)  jc=(int) fmodf(jc,dom->Gfz.jn);
if(kc<dom->Gfz.ks||kc > dom->Gfz.ke-2)  kc=(int) fmodf(kc,dom->Gfz.kn-1);
  if(ic < dom->Gfz.is) ic = ic+dom->Gfz.in;
  if(jc < dom->Gfz.js) jc = jc+dom->Gfz.jn;
  if(kc < dom->Gfz.ks) kc = kc+(dom->Gfz.kn-1);
break;
default:
break;
}

}

__device__ void periodic_grid_position(real x,real y,real z,dom_struct *dom)
{
if(x<dom->xs||x>dom->xe)  x=fmodf(x,dom->xl);
if(y<dom->ys||y>dom->ye)  y=fmodf(y,dom->yl);
if(z<dom->zs||z>dom->ze)  z=fmodf(z,dom->zl);

  if(x < dom->xs ) x = x + dom->xl;
  if(y < dom->ys ) y = y + dom->yl;
  if(z < dom->zs ) z = z + dom->zl;
}

//Gausian kernel to calculate weight coefficient of the filter
__device__ real lpt_integrate_mol(int ic,int jc,int kc,real xp,real yp,real zp, real dx,real dy,real dz,real xs,real ys,real zs, int dirc)
{

real xm,ym,zm;
real x,y,z,r2;
switch(dirc)
{
case 0:
xm = (ic-DOM_BUF+0.5) * dx + xs;
ym = (jc-DOM_BUF+0.5) * dy + ys;
zm = (kc-DOM_BUF+0.5) * dz + zs;
r2 = (xp-xm)*(xp-xm)+(yp-ym)*(yp-ym)+(zp-zm)*(zp-zm);
break;
case 1:
x =  (ic-DOM_BUF) * dx + xs;
ym = (jc-DOM_BUF+0.5) * dy + ys;
zm = (kc-DOM_BUF+0.5) * dz + zs;
r2 = (xp-x)*(xp-x)+(yp-ym)*(yp-ym)+(zp-zm)*(zp-zm);
break;
case 2:
xm = (ic-DOM_BUF+0.5) * dx + xs;
y = (jc-DOM_BUF) * dy + ys;
zm = (kc-DOM_BUF+0.5) * dz + zs;
r2 = (xp-xm)*(xp-xm)+(yp-y)*(yp-y)+(zp-zm)*(zp-zm);
break;
case 3:
xm = (ic-DOM_BUF+0.5) * dx + xs;
ym = (jc-DOM_BUF+0.5) * dy + ys;
z = (kc-DOM_BUF) * dz + zs;
r2 = (xp-xm)*(xp-xm)+(yp-ym)*(yp-ym)+(zp-z)*(zp-z);
break;
default:
printf("Wrong dirc in lpt_integrate_mol");
}

//TODO make this as defined value avaible from host and device!!!
real min_meshsize=min(min(dx,dy),dz);
//2.0f*sqrt(2.0f*log(2.0f)) = 2.355
real sig= KERNEL_WIDTH *min_meshsize/(2.0f*sqrt(2.0f*log(2.0f)));

//r should be 3 times larger than sig to make val smaller than 0.01!!
real val = exp(-r2/(2.0f*sig*sig));
return val;
}




// xp~zp is the particle location, x(1~N) is the grid face position
// if x(ip)<=xp<x(ip+1) points[pp].i=ip, this is the grid face number rather than grid center number
__global__ void lpt_localize(int npoints, point_struct *points, dom_struct *dom, BC bc)
{
 int pp =  threadIdx.x + blockIdx.x*blockDim.x;

if(pp<npoints)
{
  real ddx = 1. / dom->dx;
  real ddy = 1. / dom->dy;
  real ddz = 1. / dom->dz;

// Cartesian location of node
real  xp =  points[pp].x;
real  yp =  points[pp].y;
real  zp =  points[pp].z;

//TODO whether periodic BC for point particle need to be determined in future
periodic_grid_position(xp,yp,zp,dom);
 
  int ip,jp,kp;
//x(i)=(i-DOM_BUF)*dom->dx+dom->xs
//i=floor((x(i) - dom->xs) * ddx) + DOM_BUF;
  ip = floor((xp - dom->xs) * ddx) + DOM_BUF;//for x[i]
  jp = floor((yp - dom->ys) * ddy) + DOM_BUF;//for y[j]
  kp = floor((zp - dom->zs) * ddz) + DOM_BUF;//for z[k]

points[pp].i= ip;
points[pp].j= jp;
points[pp].k= kp;
}

}


__global__ void stress_u(real rho_f, real nu, real *u0,real *p,real *p0,real *stress, dom_struct *dom,int *flag_u, real dt, real dt0)
{
  // create shared memory
  // no reason to load pressure into shared memory, but leaving it in global
  // will require additional if statements, so keep it in shared
  __shared__ real s_u0[MAX_THREADS_DIM * MAX_THREADS_DIM];      // u back
  __shared__ real s_u1[MAX_THREADS_DIM * MAX_THREADS_DIM];      // u center
  __shared__ real s_u2[MAX_THREADS_DIM * MAX_THREADS_DIM];      // u forward

  __shared__ real s_d[MAX_THREADS_DIM * MAX_THREADS_DIM];       // diff
  
  __shared__ real grad_P[MAX_THREADS_DIM * MAX_THREADS_DIM];       // diff
  __shared__ real grad_P0[MAX_THREADS_DIM * MAX_THREADS_DIM];       // diff
 
// working constants

/*
  real a = (0.5 * dt) / (0.5 * dt0 + 0.5 * dt); //why in time??
  real ab0= 0.5 * a ;
  real ab=1. + ab0 ;
  */

//Weight for Adam-Bashforth interpolation
  real a = dt0/dt;
  a = (a + 2.)/(a + 1.);
  real ab0= a-1.0 ;
  real ab=a;

  real ddx = 1. / dom->dx;     // to limit the number of divisions needed
  real ddy = 1. / dom->dy;     // to limit the number of divisions needed
  real ddz = 1. / dom->dz;     // to limit the number of divisions needed


  // loop over u-planes
  for(int i = dom->Gfx._is; i < dom->Gfx._ie; i++) {

 // subdomain indices
    // the extra 2*blockIdx.X terms implement the necessary overlapping of
    // shared memory blocks in the subdomain
    int j = blockIdx.x*blockDim.x + threadIdx.x - 2*blockIdx.x;
    int k = blockIdx.y*blockDim.y + threadIdx.y - 2*blockIdx.y;
    // shared memory indices
    int tj = threadIdx.x;
    int tk = threadIdx.y;

 // load shared memory
    // TODO: look into the effect of removing these if statements and simply
    // allowing memory overruns for threads that don't matter for particular
    // discretizations
    // TODO: THIS CAN BE FIXED BY PADDING ALL OF THESE ARRAYS WHEN COPYING FROM
    // HOST TO DEVICE
    if((k >= dom->Gfx._ksb && k < dom->Gfx._keb)
      && (j >= dom->Gfx._jsb && j < dom->Gfx._jeb)) {
      s_u0[tj + tk*blockDim.x] = u0[(i-1) + j*dom->Gfx._s1b + k*dom->Gfx._s2b];
      s_u1[tj + tk*blockDim.x] = u0[i + j*dom->Gfx._s1b + k*dom->Gfx._s2b];
      s_u2[tj + tk*blockDim.x] = u0[(i+1) + j*dom->Gfx._s1b + k*dom->Gfx._s2b];
    }
   
    __syncthreads();


    // if off the shared memory block boundary
    if((tj > 0 && tj < blockDim.x-1) && (tk > 0 && tk < blockDim.y-1)) {
      real u011 = s_u0[tj + tk*blockDim.x];
      real u111 = s_u1[tj + tk*blockDim.x];
      real u211 = s_u2[tj + tk*blockDim.x];

      real u101 = s_u1[(tj-1) + tk*blockDim.x];
      real u121 = s_u1[(tj+1) + tk*blockDim.x];

      real u110 = s_u1[tj + (tk-1)*blockDim.x];
      real u112 = s_u1[tj + (tk+1)*blockDim.x];

 // compute diffusion term (Adams-Bashforth stepping)
      real dud1 = (u211 - u111) * ddx;
      real dud0 = (u111 - u011) * ddx;
      real ddudxx = (dud1 - dud0) * ddx;

      dud1 = (u121 - u111) * ddy;
      dud0 = (u111 - u101) * ddy;
      real ddudyy = (dud1 - dud0) * ddy;

      dud1 = (u112 - u111) * ddz;
      dud0 = (u111 - u110) * ddz;
      real ddudzz = (dud1 - dud0) * ddz;

      s_d[tj + tk*blockDim.x] = nu * (ddudxx + ddudyy + ddudzz);
/*
      grad_P[tj + tk*blockDim.x]=abs(flag_u[i + tj*dom->Gfx._s1b
        + tk*dom->Gfx._s2b])
        * ddx * (p[i + tj*dom->Gcc._s1b + tk*dom->Gcc._s2b]
        - p[(i-1) + tj*dom->Gcc._s1b + tk*dom->Gcc._s2b]);
  */    
      grad_P0[tj + tk*blockDim.x]=abs(flag_u[i + tj*dom->Gfx._s1b
        + tk*dom->Gfx._s2b])
        * ddx * (p0[i + tj*dom->Gcc._s1b + tk*dom->Gcc._s2b]
        - p0[(i-1) + tj*dom->Gcc._s1b + tk*dom->Gcc._s2b]);

    }
   // make sure all threads complete computations
    __syncthreads();

    // copy shared memory back to global
    if((k >= dom->Gfx._ks && k < dom->Gfx._ke)
      && (j >= dom->Gfx._js && j < dom->Gfx._je)
      && (tj > 0 && tj < (blockDim.x-1))
      && (tk > 0 && tk < (blockDim.y-1))) { 
//     stress[i + j*dom->Gfx._s1b + k*dom->Gfx._s2b] =rho_f* s_d[tj + tk*blockDim.x]-ab*grad_P[tj + tk*blockDim.x]+ab0*grad_P0[tj + tk*blockDim.x]-gradP_x;
     stress[i + j*dom->Gfx._s1b + k*dom->Gfx._s2b] =rho_f* s_d[tj + tk*blockDim.x]-ab*grad_P[tj + tk*blockDim.x]+ab0*grad_P0[tj + tk*blockDim.x];
// stress[i + j*dom->Gfx._s1b + k*dom->Gfx._s2b] =rho_f* s_d[tj + tk*blockDim.x]-grad_P[tj + tk*blockDim.x];
    }
  }
} 


__global__ void stress_v(real rho_f, real nu, real *v0,real *p,real *p0,real *stress, dom_struct *dom,int *flag_v, real dt, real dt0)
{
  // create shared memory
  // no reason to load pressure into shared memory, but leaving it in global
  // will require additional if statements, so keep it in shared
  __shared__ real s_v0[MAX_THREADS_DIM * MAX_THREADS_DIM];      // v back
  __shared__ real s_v1[MAX_THREADS_DIM * MAX_THREADS_DIM];      // v center
  __shared__ real s_v2[MAX_THREADS_DIM * MAX_THREADS_DIM];      // v forward

  __shared__ real s_d[MAX_THREADS_DIM * MAX_THREADS_DIM];       // diff

  __shared__ real grad_P[MAX_THREADS_DIM * MAX_THREADS_DIM];       // y-force
 __shared__ real grad_P0[MAX_THREADS_DIM * MAX_THREADS_DIM];       // diff
 
// working constants
/*  real a = (0.5 * dt) / (0.5 * dt0 + 0.5 * dt); //why in time??
  real ab0= 0.5 * a ;
  real ab=1. + ab0 ;
  */
 real a = dt0/dt;
  a = (a + 2.)/(a + 1.);
  real ab0= a-1.0 ;
  real ab=a;

  // working constants
 
  real ddx = 1. / dom->dx;     // to limit the number of divisions needed
  real ddy = 1. / dom->dy;     // to limit the number of divisions needed
  real ddz = 1. / dom->dz;     // to limit the number of divisions needed

  // loop over u-planes
  for(int j = dom->Gfy._js; j < dom->Gfy._je; j++) {
    // subdomain indices
    // the extra 2*blockIdx.X terms implement the necessary overlapping of
    // shared memory blocks in the subdomain
    int k = blockIdx.x*blockDim.x + threadIdx.x - 2*blockIdx.x;
    int i = blockIdx.y*blockDim.y + threadIdx.y - 2*blockIdx.y;
    // shared memory indices
    int tk = threadIdx.x;
    int ti = threadIdx.y;

    // load shared memory
    // TODO: look into the effect of removing these if statements and simply
    // allowing memory overruns for threads that don't matter for particular
    // discretizations
    if((i >= dom->Gfy._isb && i < dom->Gfy._ieb)
      && (k >= dom->Gfy._ksb && k < dom->Gfy._keb)) {
     
      s_v0[tk + ti*blockDim.x] = v0[i + (j-1)*dom->Gfy._s1b + k*dom->Gfy._s2b];
      s_v1[tk + ti*blockDim.x] = v0[i + j*dom->Gfy._s1b + k*dom->Gfy._s2b];
      s_v2[tk + ti*blockDim.x] = v0[i + (j+1)*dom->Gfy._s1b + k*dom->Gfy._s2b];
     }
   
    // make sure all threads complete shared memory copy
    __syncthreads();

    // compute right-hand side
    // if off the shared memory block boundary
    if((tk > 0 && tk < blockDim.x-1) && (ti > 0 && ti < blockDim.y-1)) {
      // grab the required data points for calculations
      real v101 = s_v0[tk + ti*blockDim.x];
      real v111 = s_v1[tk + ti*blockDim.x];
      real v121 = s_v2[tk + ti*blockDim.x];

      real v110 = s_v1[(tk-1) + ti*blockDim.x];
      real v112 = s_v1[(tk+1) + ti*blockDim.x];
   
      real v011 = s_v1[tk + (ti-1)*blockDim.x];
      real v211 = s_v1[tk + (ti+1)*blockDim.x];

      // compute diffusive term
      real dvd1 = (v211 - v111) * ddx;
      real dvd0 = (v111 - v011) * ddx;
      real ddvdxx = (dvd1 - dvd0) * ddx;

      dvd1 = (v121 - v111) * ddy;
      dvd0 = (v111 - v101) * ddy;
      real ddvdyy = (dvd1 - dvd0) * ddy;

      dvd1 = (v112 - v111) * ddz;
      dvd0 = (v111 - v110) * ddz;
      real ddvdzz = (dvd1 - dvd0) * ddz;

      s_d[tk + ti*blockDim.x] = nu * (ddvdxx + ddvdyy + ddvdzz);
/* 
     grad_P[tk + ti*blockDim.x]=abs(flag_v[ti + j*dom->Gfy._s1b
        + tk*dom->Gfy._s2b])
        * ddy * (p[ti + j*dom->Gcc._s1b + tk*dom->Gcc._s2b]
        - p[ti + (j-1)*dom->Gcc._s1b + tk*dom->Gcc._s2b]);
*/
      grad_P0[tk + ti*blockDim.x]=abs(flag_v[ti + j*dom->Gfy._s1b
        + tk*dom->Gfy._s2b])
        * ddy * (p0[ti + j*dom->Gcc._s1b + tk*dom->Gcc._s2b]
        - p0[ti + (j-1)*dom->Gcc._s1b + tk*dom->Gcc._s2b]);

     
	}

    // make sure all threads complete computations
    __syncthreads();

    // copy shared memory back to global
    if((i >= dom->Gfy._is && i < dom->Gfy._ie)
      && (k >= dom->Gfy._ks && k < dom->Gfy._ke)
      && (tk > 0 && tk < (blockDim.x-1))
      && (ti > 0 && ti < (blockDim.y-1))) {
  //    stress[i + j*dom->Gfy._s1b + k*dom->Gfy._s2b] = rho_f*s_d[tk + ti*blockDim.x]-ab*grad_P[tk + ti*blockDim.x]+ab0*grad_P0[tk + ti*blockDim.x]-gradP_y;
//     stress[i + j*dom->Gfy._s1b + k*dom->Gfy._s2b] = rho_f*s_d[tk + ti*blockDim.x]-grad_P[tk + ti*blockDim.x];
    stress[i + j*dom->Gfy._s1b + k*dom->Gfy._s2b] = rho_f*s_d[tk + ti*blockDim.x]-ab*grad_P[tk + ti*blockDim.x]+ab0*grad_P0[tk + ti*blockDim.x];
      }
  }
}

__global__ void stress_w(real rho_f, real nu, real *w0,real *p,real *p0,real *stress, dom_struct *dom,int *flag_w, real dt, real dt0)
{
  // create shared memory
  // no reason to load pressure into shared memory, but leaving it in global
  // will require additional if statements, so keep it in shared
  __shared__ real s_w0[MAX_THREADS_DIM * MAX_THREADS_DIM];      // w back
  __shared__ real s_w1[MAX_THREADS_DIM * MAX_THREADS_DIM];      // w center
  __shared__ real s_w2[MAX_THREADS_DIM * MAX_THREADS_DIM];      // w forward
  
  __shared__ real s_d[MAX_THREADS_DIM * MAX_THREADS_DIM];       // diff0
  
  __shared__ real grad_P[MAX_THREADS_DIM * MAX_THREADS_DIM];  // pressure gradient
  __shared__ real grad_P0[MAX_THREADS_DIM * MAX_THREADS_DIM];       // diff
 
// working constants
/*  real a = (0.5 * dt) / (0.5 * dt0 + 0.5 * dt); //why in time??
  real ab0= 0.5 * a ;
  real ab=1. + ab0 ;
  */
  real a = dt0/dt;
  a = (a + 2.)/(a + 1.);
  real ab0= a-1.0 ;
  real ab=a;

  // working constants
  real ddx = 1. / dom->dx;     // to limit the number of divisions needed
  real ddy = 1. / dom->dy;     // to limit the number of divisions needed
  real ddz = 1. / dom->dz;     // to limit the number of divisions needed

  // loop over w-planes
  for(int k = dom->Gfz._ks; k < dom->Gfz._ke; k++) {
    // subdomain indices
    // the extra 2*blockIdx.X terms implement the necessary overlapping of
    // shared memory blocks in the subdomain
    int i = blockIdx.x*blockDim.x + threadIdx.x - 2*blockIdx.x;
    int j = blockIdx.y*blockDim.y + threadIdx.y - 2*blockIdx.y;
    // shared memory indices
    int ti = threadIdx.x;
    int tj = threadIdx.y;

    // load shared memory
    // TODO: look into the effect of removing these if statements and simply
    // allowing memory overruns for threads that don't matter for particular
    // discretizations
    if((j >= dom->Gfz._jsb && j < dom->Gfz._jeb)
      && (i >= dom->Gfz._isb && i < dom->Gfz._ieb)) {
      
      s_w0[ti + tj*blockDim.x] = w0[i + j*dom->Gfz._s1b + (k-1)*dom->Gfz._s2b];
      s_w1[ti + tj*blockDim.x] = w0[i + j*dom->Gfz._s1b + k*dom->Gfz._s2b];
      s_w2[ti + tj*blockDim.x] = w0[i + j*dom->Gfz._s1b + (k+1)*dom->Gfz._s2b];
      
    }
     // make sure all threads complete shared memory copy
    __syncthreads();

    // compute right-hand side
    // if off the shared memory block boundary
    if((ti > 0 && ti < blockDim.x-1) && (tj > 0 && tj < blockDim.y-1)) {
      // grab the required data points for calculations
      real w110 = s_w0[ti + tj*blockDim.x];
      real w111 = s_w1[ti + tj*blockDim.x];
      real w112 = s_w2[ti + tj*blockDim.x];

      real w011 = s_w1[(ti-1) + tj*blockDim.x];
      real w211 = s_w1[(ti+1) + tj*blockDim.x];
   
      real w101 = s_w1[ti + (tj-1)*blockDim.x];
      real w121 = s_w1[ti + (tj+1)*blockDim.x];

      // compute diffusive term
      real dwd1 = (w211 - w111) * ddx;
      real dwd0 = (w111 - w011) * ddx;
      real ddwdxx = (dwd1 - dwd0) * ddx;

      dwd1 = (w121 - w111) * ddy;
      dwd0 = (w111 - w101) * ddy;
      real ddwdyy = (dwd1 - dwd0) * ddy;

      dwd1 = (w112 - w111) * ddz;
      dwd0 = (w111 - w110) * ddz;
      real ddwdzz = (dwd1 - dwd0) * ddz;

      s_d[ti + tj*blockDim.x] = nu * (ddwdxx + ddwdyy + ddwdzz);
/*
      grad_P[ti + tj*blockDim.x] = abs(flag_w[ti + tj*dom->Gfz._s1b
        + k*dom->Gfz._s2b])
        * ddz * (p[ti + tj*dom->Gcc._s1b + k*dom->Gcc._s2b]
        - p[ti + tj*dom->Gcc._s1b + (k-1)*dom->Gcc._s2b]);
*/	
      grad_P0[ti + tj*blockDim.x] = abs(flag_w[ti + tj*dom->Gfz._s1b
        + k*dom->Gfz._s2b])
        * ddz * (p0[ti + tj*dom->Gcc._s1b + k*dom->Gcc._s2b]
        - p0[ti + tj*dom->Gcc._s1b + (k-1)*dom->Gcc._s2b]);

 
    }

    // make sure all threads complete computations
    __syncthreads();

    // copy shared memory back to global
    if((j >= dom->Gfz._js && j < dom->Gfz._je)
      && (i >= dom->Gfz._is && i < dom->Gfz._ie)
      && (ti > 0 && ti < (blockDim.x-1))
      && (tj > 0 && tj < (blockDim.y-1))) {
//      stress[i + j*dom->Gfz._s1b + k*dom->Gfz._s2b]= rho_f* s_d[ti + tj*blockDim.x]-ab*grad_P[ti + tj*blockDim.x]+ab0*grad_P0[ti + tj*blockDim.x]-gradP_z;
//stress[i + j*dom->Gfz._s1b + k*dom->Gfz._s2b]= rho_f* s_d[ti + tj*blockDim.x]-grad_P[ti + tj*blockDim.x];
      
stress[i + j*dom->Gfz._s1b + k*dom->Gfz._s2b]= rho_f* s_d[ti + tj*blockDim.x]-ab*grad_P[ti + tj*blockDim.x]+ab0*grad_P0[ti + tj*blockDim.x];
    }
  }
}



//Interpolate cell-center flow field data A to the particle position, and get Ag at each particle position
__global__ void interpolate_point_scalar_Lag2(int npoints,real *A,real *Ag, point_struct *points, dom_struct *dom)
{
  int pp =  threadIdx.x + blockIdx.x*blockDim.x;
  real ddx = 1. / dom->dx;
  real ddy = 1. / dom->dy;
  real ddz = 1. / dom->dz;
 
if(pp<npoints)
{

real  x =  points[pp].x;
real  y =  points[pp].y;
real  z =  points[pp].z;
//real Ap=points[pp].mdot;
 
//TODO threat BC in the future
//periodic_grid_position(x,y,z,dom);

 __syncthreads();
int i,j,k;   
real x1,x2,y1,y2,z1,z2;
int ic,jc,kc;
real wx[2],wy[2],wz[2];
real weight[2][2][2];

  i = floor((x - dom->xs) * ddx- 0.5) + DOM_BUF;//for xm[i]
  j = floor((y - dom->ys) * ddy- 0.5) + DOM_BUF;//for ym[j]
  k = floor((z - dom->zs) * ddz- 0.5) + DOM_BUF;//for zm[k]

//periodic_grid_index(i,j,k,dom,0);

  x1 = (i-0.5) * dom->dx + dom->xs;
  x2 = (i+0.5) * dom->dx + dom->xs;
  y1=  (j-0.5) * dom->dy + dom->ys;
  y2=  (j+0.5) * dom->dy + dom->ys;
  z1=  (k-0.5) * dom->dz + dom->zs;
  z2=  (k+0.5) * dom->dz + dom->zs;

//2nd order lagragian interpolation
 wx[0]=(x2-x)/(x2-x1);
 wx[1]=(x-x1)/(x2-x1);
 wy[0]=(y2-y)/(y2-y1);
 wy[1]=(y-y1)/(y2-y1);
 wz[0]=(z2-z)/(z2-z1);
 wz[1]=(z-z1)/(z2-z1);

real buf=0.f;
//Normalize the weight
 for(int kk = 0; kk < 2; kk++) 
   for(int jj = 0; jj < 2; jj++) 
    for(int ii = 0; ii < 2; ii++) 
      { 
	weight[ii][jj][kk]=wx[ii]*wy[jj]*wz[kk];	
	buf+=weight[ii][jj][kk];
	}

//Normalize
if(fabs(buf-1)>EPSILON)
{
 for(int kk = 0; kk < 2; kk++) 
   for(int jj = 0; jj < 2; jj++) 
    for(int ii = 0; ii < 2; ii++) 
	weight[ii][jj][kk]=weight[ii][jj][kk]/buf;	
}

Ag[pp]=0;
//Add them to the scalar field source
 for(int kk = 0; kk < 2; kk++) 
   for(int jj = 0; jj < 2; jj++) 
    for(int ii = 0; ii < 2; ii++) 
      { 
  ic=i+ii;jc=j+jj;kc=k+kk;
periodic_grid_index(ic,jc,kc,dom,0);
  Ag[pp]+=weight[ii][jj][kk]*A[ic +jc*dom->Gcc.s1b + kc*dom->Gcc.s2b];
 }

//TODO add mask infomation as in lpt_mollify_sc in lpt_interpolator.f90
  }
}

//Bilinear interpolation!  This skeme is accurate enough, which has been tested by oscllating flows.
//TODO treat boudary specification,such as mask information
__global__ void interpolate_point_vel_Lag2(real *u, real *v, real *w,int npoints,real rho_f, real nu,real *ug,real *vg,real *wg, point_struct *points, dom_struct *dom, BC bc)
{
  // int node = threadIdx.x;
  int pp =  threadIdx.x + blockIdx.x*blockDim.x;
  real ddx = 1. / dom->dx;
  real ddy = 1. / dom->dy;
  real ddz = 1. / dom->dz;


if(pp<npoints)
{
// Cartesian location of node
real  x =  points[pp].x;
real  y =  points[pp].y;
real  z =  points[pp].z;

// periodic_grid_position(x,y,z,dom);

  __syncthreads();

int i,j,k;   
real x1,x2,y1,y2,z1,z2;
int ic,jc,kc;
real wx[2],wy[2],wz[2];
real weight[2][2][2];

  // interpolate velocities
  // interpolate u-velocity
  i = floor((x - dom->xs) * ddx) + DOM_BUF; 	//for x[i]
  j = floor((y - dom->ys) * ddy- 0.5) + DOM_BUF;//for ym[j]
  k = floor((z - dom->zs) * ddz- 0.5) + DOM_BUF;//for zm[k]
//periodic_grid_index(ic,jc,kc,dom,1);

  x1 = (i-DOM_BUF) * dom->dx + dom->xs;
  x2 = (i+1-DOM_BUF) * dom->dx + dom->xs;
  y1=  (j-0.5) * dom->dy + dom->ys;
  y2=  (j+0.5) * dom->dy + dom->ys;
  z1=  (k-0.5) * dom->dz + dom->zs;
  z2=  (k+0.5) * dom->dz + dom->zs;

 
 wx[0]=(x2-x)/(x2-x1);
 wx[1]=(x-x1)/(x2-x1);
 wy[0]=(y2-y)/(y2-y1);
 wy[1]=(y-y1)/(y2-y1);
 wz[0]=(z2-z)/(z2-z1);
 wz[1]=(z-z1)/(z2-z1);

ug[pp]=0;
  for(int kk = 0; kk < 2; kk++) 
   for(int jj = 0; jj < 2; jj++) 
    for(int ii = 0; ii < 2; ii++) 
      { 
	
weight[ii][jj][kk]=wx[ii]*wy[jj]*wz[kk];	
ic=i+ii;
jc=j+jj;
kc=k+kk;
periodic_grid_index(ic,jc,kc,dom,1);
	ug[pp]+=weight[ii][jj][kk]*u[ic +jc*dom->Gfx.s1b + kc*dom->Gfx.s2b];
	}

  // interpolate V-velocity
  i = floor((x - dom->xs) * ddx- 0.5) + DOM_BUF;
  j = floor((y - dom->ys) * ddy) + DOM_BUF;
  k = floor((z - dom->zs) * ddz- 0.5) + DOM_BUF;

//periodic_grid_index(ic,jc,kc,dom,2);

  x1 = (i-0.5) * dom->dx + dom->xs;
  x2 = (i+0.5) * dom->dx + dom->xs;
  y1=  (j-DOM_BUF) * dom->dy + dom->ys;
  y2=  (j+1-DOM_BUF) * dom->dy + dom->ys;
  z1=  (k-0.5) * dom->dz + dom->zs;
  z2=  (k+0.5) * dom->dz + dom->zs;


 wx[0]=(x2-x)/(x2-x1);
 wx[1]=(x-x1)/(x2-x1);
 wy[0]=(y2-y)/(y2-y1);
 wy[1]=(y-y1)/(y2-y1);
 wz[0]=(z2-z)/(z2-z1);
 wz[1]=(z-z1)/(z2-z1);

vg[pp]=0;
  for(int kk = 0; kk < 2; kk++) 
   for(int jj = 0; jj < 2; jj++) 
    for(int ii = 0; ii < 2; ii++) 
      { 
	weight[ii][jj][kk]=wx[ii]*wy[jj]*wz[kk];
ic=i+ii;
jc=j+jj;
kc=k+kk;
periodic_grid_index(ic,jc,kc,dom,2);
	vg[pp]+=weight[ii][jj][kk]*v[ic +jc*dom->Gfy.s1b + kc*dom->Gfy.s2b];
	}


  // interpolate W-velocity
  i = floor((x - dom->xs) * ddx- 0.5) + DOM_BUF;
  j = floor((y - dom->ys) * ddy- 0.5) + DOM_BUF;
  k = floor((z - dom->zs) * ddz) + DOM_BUF;
//periodic_grid_index(ic,jc,kc,dom,3);

  x1 = (i-0.5) * dom->dx + dom->xs;
  x2 = (i+0.5) * dom->dx + dom->xs;
  y1=  (j-0.5) * dom->dy + dom->ys;
  y2=  (j+0.5) * dom->dy + dom->ys;
  z1=  (k-DOM_BUF) * dom->dz + dom->zs;
  z2=  (k+1-DOM_BUF) * dom->dz + dom->zs;


 wx[0]=(x2-x)/(x2-x1);
 wx[1]=(x-x1)/(x2-x1);
 wy[0]=(y2-y)/(y2-y1);
 wy[1]=(y-y1)/(y2-y1);
 wz[0]=(z2-z)/(z2-z1);
 wz[1]=(z-z1)/(z2-z1);

wg[pp]=0;

  for(int kk = 0; kk < 2; kk++) 
   for(int jj = 0; jj < 2; jj++) 
    for(int ii = 0; ii < 2; ii++) 
      { 
	weight[ii][jj][kk]=wx[ii]*wy[jj]*wz[kk];
ic=i+ii;
jc=j+jj;
kc=k+kk;
periodic_grid_index(ic,jc,kc,dom,3);
	wg[pp]+=weight[ii][jj][kk]*w[ic + jc*dom->Gfz.s1b + kc*dom->Gfz.s2b];
	}
 }

}


__global__ void point_interp_init(int npoints,point_struct *points,real *ug,real *vg,real *wg,real *lpt_stress_u,real *lpt_stress_v,real *lpt_stress_w,real *scg)
{
 int pp = threadIdx.x + blockIdx.x*blockDim.x;
  if(pp < npoints) {
//real rad=  points[pp].r;
ug[pp]=0;
vg[pp]=0;
wg[pp]=0;

lpt_stress_u[pp]=0;
lpt_stress_v[pp]=0;
lpt_stress_w[pp]=0;

scg[pp]=0;
//volPoint[pp]=PI*4/3*rad*rad*rad;
}
}

__global__ void drag_points(point_struct *points, int npoints,
real *ug,real *vg,real *wg,
real *lpt_stress_u,real *lpt_stress_v,real *lpt_stress_w,real *scg,
real rho_f,real mu, g_struct g,gradP_struct gradP,
real C_add,real C_stress,real C_drag,
real sc_eq,real DIFF)
//gradP serve as bodyforce for the time being
{
  int pp = threadIdx.x + blockIdx.x*blockDim.x;

  if(pp < npoints) {
real up=points[pp].u;
real vp=points[pp].v;
real wp=points[pp].w;
real dia=2*points[pp].r;
real rhod=points[pp].rho;//rhod is the particle density

//particle interaction force
real iFx=points[pp].iFx;
real iFy=points[pp].iFy;
real iFz=points[pp].iFz;

//fluid velocity at the particle position
real uf=ug[pp];
real vf=vg[pp];
real wf=wg[pp];

//Fluid stress at the particle position
real stress_x=lpt_stress_u[pp];
real stress_y=lpt_stress_v[pp];
real stress_z=lpt_stress_w[pp];


//realative velocity between point_particle and fluid
real ur=sqrt((up-uf)*(up-uf)+(vp-vf)*(vp-vf)+(wp-wf)*(wp-wf));
real nu=mu/rho_f;

//Particle Reynolds number
real Rep=ur*dia/nu+EPSILON;


//Based on hp*dp/D=2+0.6Re_p^0.5 Sc^{1/3}, ref eq (22) in Oresta&&Prosperetti(2014)
real Nu =2+0.6*sqrt(Rep)*powf(nu/DIFF,1.0/3.0);

//Nu=2;
real hp =Nu*DIFF/dia;
/*
real hh =(0.6*sqrt(Rep)*powf(nu/DIFF,1.0/3.0));
int n3=5;
int n2=1;
real r=n3*DIFF/PI/2;
real strength=r/n2;
if(ur>0.5*strength) printf("\nhh %f %f %f %f\n",hh,Rep,ur,DIFF);
*/

//Modification to stokes theory when Rep>1
real F = 1.0f+0.15f*powf(Rep,0.687f); 
//real F = 1.0f; 
real taud = rhod*dia*dia/(18.0f*mu);
real itau=F/taud;

real volume=1./6. * PI * dia*dia*dia;

//Including soluble part:rhod*volume  and insoluble part: ms
real mp =  rhod *volume + points[pp].ms;
//fluid mass in particle volume
real mf =  rho_f *volume;
real msdot=points[pp].msdot;
real gammar=mp/mf;

//drag force on particle
real drag_x=(uf-up)*itau*mp;
real drag_y=(vf-vp)*itau*mp;
real drag_z=(wf-wp)*itau*mp;


//Total add mass,  -gradP.x~z are the body force on fluid apart from gravity
real add_x=(stress_x/rho_f-gradP.x)*mf;
real add_y=(stress_y/rho_f-gradP.y)*mf;
real add_z=(stress_z/rho_f-gradP.z)*mf;

//Store the fluid force on particle including add mass, drag, fluid force and gravity
//default C_add=0.5; C_stress=1; C_drag=1;
real Fx=(C_add*add_x+C_stress*stress_x*volume+C_drag*drag_x)+(mp-mf)*g.x;
real Fy=(C_add*add_y+C_stress*stress_y*volume+C_drag*drag_y)+(mp-mf)*g.y;
real Fz=(C_add*add_z+C_stress*stress_z*volume+C_drag*drag_z)+(mp-mf)*g.z;

//Store the temp particle acceleration, also including the particle soluble mass change  in the last term!
real udot =(Fx+ iFx -up*msdot)/mp;
real vdot =(Fy+ iFy -vp*msdot)/mp;
real wdot =(Fz+ iFz -wp*msdot)/mp;

//acount for added mass effect since it appears also on the left handside of particle governing equation
if(fabs(C_add-0.5)<EPSILON)
{
      udot = udot/(1+C_add/gammar);
      vdot = vdot/(1+C_add/gammar);
      wdot = wdot/(1+C_add/gammar);
}

//particle acceleration
      points[pp].udot = udot;
      points[pp].vdot = vdot;
      points[pp].wdot = wdot;


/*
Store the fluid force on particle including add mass, drag, fluid force and gravity
added mass effect on particle-fluid force interaction
gravity is not implemented on fluid, but we need to add -mf*g to reaction force
F=d(mp*u)/dt -mp*g, according to eq 6 from dropDiffusionForceImpleBlue.pdf
*/
points[pp].Fx=Fx-C_add*udot*mf-mp*g.x;
points[pp].Fy=Fy-C_add*vdot*mf-mp*g.y;
points[pp].Fz=Fz-C_add*wdot*mf-mp*g.z;


/*
Exchange rate of soluble mass into scalar field
dms/dt = pi *dp^2*hp*(rho_s-rho_{sat})   ref eq (20) in Oresta&&Prosperetti(2014)
*/
if(points[pp].ms>0)  points[pp].msdot= PI*dia*dia*hp*(scg[pp]- sc_eq);
else   points[pp].msdot= 0;

//printf("\nmsdot %f %f %f %f\n",points[pp].msdot,hp,scg[pp],sc_eq);

  }
}



//update point velocity -> 1st step of Eulerian prediction
__global__ void move_points_a(point_struct *points, int npoints,
  real dt)
{

  int pp = threadIdx.x + blockIdx.x*blockDim.x; // point_point_particle number
  
  //real m = 4./3. * PI * points[pp].rho * points[pp].r*points[pp].r*points[pp].r;
  //real dT = dt / dt0;

  if(pp < npoints) {
    // update position
      points[pp].x = points[pp].x0 + 0.5*points[pp].u * dt;
      points[pp].y = points[pp].y0 + 0.5*points[pp].v * dt;
      points[pp].z = points[pp].z0 + 0.5*points[pp].w * dt;

     // update linear velocities
      points[pp].u = points[pp].u0 + 0.5*points[pp].udot * dt;
      points[pp].v = points[pp].v0 + 0.5*points[pp].vdot * dt;
      points[pp].w = points[pp].w0 + 0.5*points[pp].wdot * dt;

     // update soluble mass
      points[pp].ms = points[pp].ms0 + 0.5*points[pp].msdot * dt;
  }
}

//update point velocity -> 2nd step of Eulerian prediction
__global__ void move_points_b(dom_struct *dom,point_struct *points, int npoints,
  real dt)
{

  int pp = threadIdx.x + blockIdx.x*blockDim.x; // point_point_particle number
  
  //real m = 4./3. * PI * points[pp].rho * points[pp].r*points[pp].r*points[pp].r;
  //real dT = dt / dt0;

  if(pp < npoints) {
    // update position
      points[pp].x = points[pp].x0 + points[pp].u * dt;
      points[pp].y = points[pp].y0 + points[pp].v * dt;
      points[pp].z = points[pp].z0 + points[pp].w * dt;

     // update linear velocities
      points[pp].u = points[pp].u0 + points[pp].udot * dt;
      points[pp].v = points[pp].v0 + points[pp].vdot * dt;
      points[pp].w = points[pp].w0 + points[pp].wdot * dt;

     // update soluble mass
      points[pp].ms = points[pp].ms0 + points[pp].msdot * dt;
  

//TODO periodic BC for particles, may need to change in future
periodic_grid_position(points[pp].x,points[pp].y,points[pp].z,dom);

//update old values
      points[pp].x0 = points[pp].x;
      points[pp].y0 = points[pp].y;
      points[pp].z0 = points[pp].z;

      points[pp].u0 = points[pp].u;
      points[pp].v0 = points[pp].v;
      points[pp].w0 = points[pp].w;

      points[pp].ms0 = points[pp].ms;
  }
}







__global__ void reset_flag_u(int *flag_u, dom_struct *dom, BC bc)
{
  int i;    // iterator
  int tj = blockDim.x*blockIdx.x + threadIdx.x;
  int tk = blockDim.y*blockIdx.y + threadIdx.y;

  // flag everything as fluid
  if((tj < dom->Gfx._jnb) && (tk < dom->Gfx._knb)) {
    for(i = dom->Gfx._isb; i < dom->Gfx._ieb; i++) {
      flag_u[i + tj*dom->Gfx._s1b + tk*dom->Gfx._s2b] = 1;
    }
  }

  __syncthreads();

  // flag external boundaries
  if((tj < dom->Gfx._jnb) && (tk < dom->Gfx._knb)) {
    if(bc.uW != PERIODIC)
      flag_u[dom->Gfx._is + tj*dom->Gfx._s1b + tk*dom->Gfx._s2b] = 0;
    if(bc.uE != PERIODIC)
      flag_u[dom->Gfx._ie-1 + tj*dom->Gfx._s1b + tk*dom->Gfx._s2b] = 0;
  }
}

__global__ void reset_flag_v(int *flag_v, dom_struct *dom, BC bc)
{
  int j;    // iterator
  int tk = blockDim.x*blockIdx.x + threadIdx.x;
  int ti = blockDim.y*blockIdx.y + threadIdx.y;

  // flag everything as fluid
  if((tk < dom->Gfy._knb) && (ti < dom->Gfy._inb)) {
    for(j = dom->Gfy._jsb; j < dom->Gfy._jeb; j++) {
      flag_v[ti + j*dom->Gfy._s1b + tk*dom->Gfy._s2b] = 1;
    }
  }

  __syncthreads();

  // flag external boundaries
  if((tk < dom->Gfy._knb) && (ti < dom->Gfy._inb)) {
    if(bc.vS != PERIODIC)
      flag_v[ti + dom->Gfy._js*dom->Gfy._s1b + tk*dom->Gfy._s2b] = 0;
    if(bc.vN != PERIODIC)
      flag_v[ti + (dom->Gfy._je-1)*dom->Gfy._s1b + tk*dom->Gfy._s2b] = 0;
  }
}

__global__ void reset_flag_w(int *flag_w, dom_struct *dom, BC bc)
{
  int k;    // iterator
  int ti = blockDim.x*blockIdx.x + threadIdx.x;
  int tj = blockDim.y*blockIdx.y + threadIdx.y;

  // flag everything as fluid
  if((ti < dom->Gfz._inb) && (tj < dom->Gfz._jnb)) {
    for(k = dom->Gfz._ksb; k < dom->Gfz._keb; k++) {
      flag_w[ti + tj*dom->Gfz._s1b + k*dom->Gfz._s2b] = 1;
    }
  }

  __syncthreads();

  // flag external boundaries
  if((ti < dom->Gfz._inb) && (tj < dom->Gfz._jnb)) {
    if(bc.wB != PERIODIC)
      flag_w[ti + tj*dom->Gfz._s1b + dom->Gfz._ks*dom->Gfz._s2b] = 0;
    if(bc.wT != PERIODIC)
      flag_w[ti + tj*dom->Gfz._s1b + (dom->Gfz._ke-1)*dom->Gfz._s2b] = 0;
  }
}


__global__ void cage_flag_u_periodic_W(int *flag_u, dom_struct *dom)
{
  int tj = blockDim.x*blockIdx.x + threadIdx.x;
  int tk = blockDim.y*blockIdx.y + threadIdx.y;

  if(tj < dom->Gfx.jnb && tk < dom->Gfx.knb) {
    flag_u[dom->Gfx.isb + tj*dom->Gfx.s1b + tk*dom->Gfx.s2b]
      = flag_u[(dom->Gfx.ie-2) + tj*dom->Gfx.s1b + tk*dom->Gfx.s2b];
  }
}

__global__ void cage_flag_u_periodic_E(int *flag_u, dom_struct *dom)
{
  int tj = blockDim.x*blockIdx.x + threadIdx.x;
  int tk = blockDim.y*blockIdx.y + threadIdx.y;

  if(tj < dom->Gfx.jnb && tk < dom->Gfx.knb) {
    flag_u[(dom->Gfx.ieb-1) + tj*dom->Gfx.s1b + tk*dom->Gfx.s2b]
      = flag_u[(dom->Gfx.is+1) + tj*dom->Gfx.s1b + tk*dom->Gfx.s2b];
  }
}

__global__ void cage_flag_u_periodic_S(int *flag_u, dom_struct *dom)
{
  int tk = blockDim.x*blockIdx.x + threadIdx.x;
  int ti = blockDim.y*blockIdx.y + threadIdx.y;

  if(tk < dom->Gfx.knb && ti < dom->Gfx.inb) {
    flag_u[ti + dom->Gfx.jsb*dom->Gfx.s1b + tk*dom->Gfx.s2b]
      = flag_u[ti + (dom->Gfx.je-1)*dom->Gfx.s1b + tk*dom->Gfx.s2b];
  }
}

__global__ void cage_flag_u_periodic_N(int *flag_u, dom_struct *dom)
{
  int tk = blockDim.x*blockIdx.x + threadIdx.x;
  int ti = blockDim.y*blockIdx.y + threadIdx.y;

  if(tk < dom->Gfx.knb && ti < dom->Gfx.inb) {
    flag_u[ti + (dom->Gfx.jeb-1)*dom->Gfx.s1b + tk*dom->Gfx.s2b]
      = flag_u[ti + dom->Gfx.js*dom->Gfx.s1b + tk*dom->Gfx.s2b];
  }
}

__global__ void cage_flag_u_periodic_B(int *flag_u, dom_struct *dom)
{
  int ti = blockDim.x*blockIdx.x + threadIdx.x;
  int tj = blockDim.y*blockIdx.y + threadIdx.y;

  if(ti < dom->Gfx.inb && tj < dom->Gfx.jnb) {
    flag_u[ti + tj*dom->Gfx.s1b + dom->Gfx.ksb*dom->Gfx.s2b]
      = flag_u[ti + tj*dom->Gfx.s1b + (dom->Gfx.ke-1)*dom->Gfx.s2b];
  }
}

__global__ void cage_flag_u_periodic_T(int *flag_u, dom_struct *dom)
{
  int ti = blockDim.x*blockIdx.x + threadIdx.x;
  int tj = blockDim.y*blockIdx.y + threadIdx.y;

  if(ti < dom->Gfx.inb && tj < dom->Gfx.jnb) {
    flag_u[ti + tj*dom->Gfx.s1b + (dom->Gfx.keb-1)*dom->Gfx.s2b]
      = flag_u[ti + tj*dom->Gfx.s1b + dom->Gfx.ks*dom->Gfx.s2b];
  }
}

__global__ void cage_flag_v_periodic_W(int *flag_v, dom_struct *dom)
{
  int tj = blockDim.x*blockIdx.x + threadIdx.x;
  int tk = blockDim.y*blockIdx.y + threadIdx.y;

  if(tj < dom->Gfy.jnb && tk < dom->Gfy.knb) {
    flag_v[dom->Gfy.isb + tj*dom->Gfy.s1b + tk*dom->Gfy.s2b]
      = flag_v[(dom->Gfy.ie-1) + tj*dom->Gfy.s1b + tk*dom->Gfy.s2b];
  }
}

__global__ void cage_flag_v_periodic_E(int *flag_v, dom_struct *dom)
{
  int tj = blockDim.x*blockIdx.x + threadIdx.x;
  int tk = blockDim.y*blockIdx.y + threadIdx.y;

  if(tj < dom->Gfy.jnb && tk < dom->Gfy.knb) {
    flag_v[(dom->Gfy.ieb-1) + tj*dom->Gfy.s1b + tk*dom->Gfy.s2b]
      = flag_v[dom->Gfy.is + tj*dom->Gfy.s1b + tk*dom->Gfy.s2b];
  }
}

__global__ void cage_flag_v_periodic_S(int *flag_v, dom_struct *dom)
{
  int tk = blockDim.x*blockIdx.x + threadIdx.x;
  int ti = blockDim.y*blockIdx.y + threadIdx.y;

  if(tk < dom->Gfy.knb && ti < dom->Gfy.inb) {
    flag_v[ti + dom->Gfy.jsb*dom->Gfy.s1b + tk*dom->Gfy.s2b]
      = flag_v[ti + (dom->Gfy.je-2)*dom->Gfy.s1b + tk*dom->Gfy.s2b];
  }
}

__global__ void cage_flag_v_periodic_N(int *flag_v, dom_struct *dom)
{
  int tk = blockDim.x*blockIdx.x + threadIdx.x;
  int ti = blockDim.y*blockIdx.y + threadIdx.y;

  if(tk < dom->Gfy.knb && ti < dom->Gfy.inb) {
    flag_v[ti + (dom->Gfy.jeb-1)*dom->Gfy.s1b + tk*dom->Gfy.s2b]
      = flag_v[ti + (dom->Gfy.js+1)*dom->Gfy.s1b + tk*dom->Gfy.s2b];
  }
}

__global__ void cage_flag_v_periodic_B(int *flag_v, dom_struct *dom)
{
  int ti = blockDim.x*blockIdx.x + threadIdx.x;
  int tj = blockDim.y*blockIdx.y + threadIdx.y;

  if(ti < dom->Gfy.inb && tj < dom->Gfy.jnb) {
    flag_v[ti + tj*dom->Gfy.s1b + dom->Gfy.ksb*dom->Gfy.s2b]
      = flag_v[ti + tj*dom->Gfy.s1b + (dom->Gfy.ke-1)*dom->Gfy.s2b];
  }
}

__global__ void cage_flag_v_periodic_T(int *flag_v, dom_struct *dom)
{
  int ti = blockDim.x*blockIdx.x + threadIdx.x;
  int tj = blockDim.y*blockIdx.y + threadIdx.y;

  if(ti < dom->Gfy.inb && tj < dom->Gfy.jnb) {
    flag_v[ti + tj*dom->Gfy.s1b + (dom->Gfy.keb-1)*dom->Gfy.s2b]
      = flag_v[ti + tj*dom->Gfy.s1b + dom->Gfy.ks*dom->Gfy.s2b];
  }
}

__global__ void cage_flag_w_periodic_W(int *flag_w, dom_struct *dom)
{
  int tj = blockDim.x*blockIdx.x + threadIdx.x;
  int tk = blockDim.y*blockIdx.y + threadIdx.y;

  if(tj < dom->Gfz.jnb && tk < dom->Gfz.knb) {
    flag_w[dom->Gfz.isb + tj*dom->Gfz.s1b + tk*dom->Gfz.s2b]
      = flag_w[(dom->Gfz.ie-1)+ tj*dom->Gfz.s1b + tk*dom->Gfz.s2b];
  }
}

__global__ void cage_flag_w_periodic_E(int *flag_w, dom_struct *dom)
{
  int tj = blockDim.x*blockIdx.x + threadIdx.x;
  int tk = blockDim.y*blockIdx.y + threadIdx.y;

  if(tj < dom->Gfz.jnb && tk < dom->Gfz.knb) {
    flag_w[(dom->Gfz.ieb-1) + tj*dom->Gfz.s1b + tk*dom->Gfz.s2b]
      = flag_w[dom->Gfz.is + tj*dom->Gfz.s1b + tk*dom->Gfz.s2b];
  }
}

__global__ void cage_flag_w_periodic_S(int *flag_w, dom_struct *dom)
{
  int tk = blockDim.x*blockIdx.x + threadIdx.x;
  int ti = blockDim.y*blockIdx.y + threadIdx.y;

  if(tk < dom->Gfz.knb && ti < dom->Gfz.inb) {
    flag_w[ti + dom->Gfz.jsb*dom->Gfz.s1b + tk*dom->Gfz.s2b]
      = flag_w[ti + (dom->Gfz.je-1)*dom->Gfz.s1b + tk*dom->Gfz.s2b];
  }
}

__global__ void cage_flag_w_periodic_N(int *flag_w, dom_struct *dom)
{
  int tk = blockDim.x*blockIdx.x + threadIdx.x;
  int ti = blockDim.y*blockIdx.y + threadIdx.y;

  if(tk < dom->Gfz.knb && ti < dom->Gfz.inb) {
    flag_w[ti + (dom->Gfz.jeb-1)*dom->Gfz.s1b + tk*dom->Gfz.s2b]
      = flag_w[ti + dom->Gfz.js*dom->Gfz.s1b + tk*dom->Gfz.s2b];
  }
}

__global__ void cage_flag_w_periodic_B(int *flag_w, dom_struct *dom)
{
  int ti = blockDim.x*blockIdx.x + threadIdx.x;
  int tj = blockDim.y*blockIdx.y + threadIdx.y;

  if(ti < dom->Gfz.inb && tj < dom->Gfz.jnb) {
    flag_w[ti + tj*dom->Gfz.s1b + dom->Gfz.ksb*dom->Gfz.s2b]
      = flag_w[ti + tj*dom->Gfz.s1b + (dom->Gfz.ke-2)*dom->Gfz.s2b];
  }
}

__global__ void cage_flag_w_periodic_T(int *flag_w, dom_struct *dom)
{
  int ti = blockDim.x*blockIdx.x + threadIdx.x;
  int tj = blockDim.y*blockIdx.y + threadIdx.y;

  if(ti < dom->Gfz.inb && tj < dom->Gfz.jnb) {
    flag_w[ti + tj*dom->Gfz.s1b + (dom->Gfz.keb-1)*dom->Gfz.s2b]
      = flag_w[ti + tj*dom->Gfz.s1b + (dom->Gfz.ks+1)*dom->Gfz.s2b];
  }
}





